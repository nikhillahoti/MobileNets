
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void executeFirstLayer_partA(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;

    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (blockIdx.y * 32 * 114)    // Position in the grid row-wise
                        + (blockIdx.z * 32)          // Position in the grid column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((blockIdx.y * 32 * 225) * stride) // Position in the grid row-wise
                       + (blockIdx.z * 32 * stride)         // Position in the grid column-wise
                       + (threadIdx.x * 225 * stride )
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    if(Z > 6.0)
      Z = 6.0;

    Layer2_Neurons_GPU[output_Position] = Z;
}


__global__ void executeFirstLayer_partB(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;

    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (blockIdx.y * 16 * 114 + 96)  // Position in the grid row-wise and there is no column-wise position
                        + (threadIdx.x * 114)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((blockIdx.y * 16 * 225) * stride) + (96 * stride) // Position in the grid row-wise and column-wise
                       + (threadIdx.x * 225 * stride)
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    if(Z > 6.0)
      Z = 6.0;

    Layer2_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFirstLayer_partC(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;

    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (96 * 114)                    // Position in the grid row-wise as row is last
                        + (blockIdx.y * 16)             // Position in the grid column-wise
                        + (threadIdx.x * 114)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((96 * 225) * stride)
                       + (blockIdx.y * 16 * stride)     // Position in the grid row-wise and column-wise
                       + (threadIdx.x * 225 * stride)
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    if(Z > 6.0)
      Z = 6.0;

    Layer2_Neurons_GPU[output_Position] = Z;
}
