
#include <hip/hip_runtime.h>
#include <stdio.h>

/*  ************************************************** FIRST LAYER START ********************************************************* */
__global__ void executeFirstLayer_partA(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;
    int outputOffset = 115;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (blockIdx.y * 32 * 114)    // Position in the grid row-wise
                        + (blockIdx.z * 32)          // Position in the grid column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((blockIdx.y * 32 * 225) * stride) // Position in the grid row-wise
                       + (blockIdx.z * 32 * stride)         // Position in the grid column-wise
                       + (threadIdx.x * 225 * stride )
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer2_Neurons_GPU[output_Position + outputOffset] = Z;
}

__global__ void executeFirstLayer_partB(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;
    int outputOffset = 115;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (blockIdx.y * 16 * 114 + 96)  // Position in the grid row-wise and there is no column-wise position
                        + (threadIdx.x * 114)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((blockIdx.y * 16 * 225) * stride) + (96 * stride) // Position in the grid row-wise and column-wise
                       + (threadIdx.x * 225 * stride)
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer2_Neurons_GPU[output_Position + outputOffset] = Z;
}

__global__ void executeFirstLayer_partC(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;
    int outputOffset = 115;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (96 * 114)                    // Position in the grid row-wise as row is last
                        + (blockIdx.y * 16)             // Position in the grid column-wise
                        + (threadIdx.x * 114)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((96 * 225) * stride)
                       + (blockIdx.y * 16 * stride)     // Position in the grid row-wise and column-wise
                       + (threadIdx.x * 225 * stride)
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer2_Neurons_GPU[output_Position + outputOffset] = Z;
}
/*  ************************************************** FIRST LAYER END ************************************************************ */

/*  ************************************************** SECOND LAYER START ********************************************************* */
__global__ void executeSecondLayer_partA(double *Layer2_Neurons_GPU,
                            double *Layer2_Weights_GPU,
                            double *Layer3_Neurons_GPU,
                            double *Layer2_Mean_GPU,
                            double *Layer2_StanDev_GPU,
                            double *Layer2_Gamma_GPU,
                            double *Layer2_Beta_GPU
                        )
{
	double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 112 * 112)   // channel to work with
                        + (blockIdx.y * 32 * 112)    // Position in the grid row-wise
                        + (blockIdx.z * 32)          // Position in the grid column-wise
                        + (threadIdx.x * 112)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (blockIdx.y * 32 * 114) // Position in the grid row-wise
                       + (blockIdx.z * 32)         // Position in the grid column-wise
                       + (threadIdx.x * 114)
                       + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114)] * Layer2_Weights_GPU[weight_Position + (row * 3)])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 1] * Layer2_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 2] * Layer2_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer2_Mean_GPU[filter_number]) / Layer2_StanDev_GPU[filter_number];
    Z = (Z * Layer2_Gamma_GPU[filter_number]) + Layer2_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer3_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSecondLayer_partB(double *Layer2_Neurons_GPU,
                                    double *Layer2_Weights_GPU,
                                    double *Layer3_Neurons_GPU,
                                    double *Layer2_Mean_GPU,
                                    double *Layer2_StanDev_GPU,
                                    double *Layer2_Gamma_GPU,
                                    double *Layer2_Beta_GPU
                                )
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 112 * 112)   // channel to work with
                        + (blockIdx.y * 16 * 112 + 96)  // Position in the grid row-wise and there is no column-wise position
                        + (threadIdx.x * 112)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position  = (blockIdx.y * 16 * 114) 
                        + (96) // Position in the grid row-wise and column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114)] * Layer2_Weights_GPU[weight_Position + (row * 3)])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 1] * Layer2_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 2] * Layer2_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer2_Mean_GPU[filter_number]) / Layer2_StanDev_GPU[filter_number];
    Z = (Z * Layer2_Gamma_GPU[filter_number]) + Layer2_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 
    
    Layer3_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSecondLayer_partC(double *Layer2_Neurons_GPU,
                                    double *Layer2_Weights_GPU,
                                    double *Layer3_Neurons_GPU,
                                    double *Layer2_Mean_GPU,
                                    double *Layer2_StanDev_GPU,
                                    double *Layer2_Gamma_GPU,
                                    double *Layer2_Beta_GPU
                                )
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 112 * 112)   // channel to work with
                        + (96 * 112)                    // Position in the grid row-wise as row is last
                        + (blockIdx.y * 16)             // Position in the grid column-wise
                        + (threadIdx.x * 112)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (96 * 114)
                        + (blockIdx.y * 16)     // Position in the grid row-wise and column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114)] * Layer2_Weights_GPU[weight_Position + (row * 3)])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 1] * Layer2_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 2] * Layer2_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer2_Mean_GPU[filter_number]) / Layer2_StanDev_GPU[filter_number];
    Z = (Z * Layer2_Gamma_GPU[filter_number]) + Layer2_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer3_Neurons_GPU[output_Position] = Z;
}
/*  ************************************************** SECOND LAYER END ********************************************************* */

/*  ************************************************** THIRD LAYER START ******************************************************** */
__global__ void executeThirdLayer_partA(double *Layer3_Neurons_GPU,
    double *Layer3_Weights_GPU,
    double *Layer4_Neurons_GPU,
    double *Layer3_Mean_GPU,
    double *Layer3_StanDev_GPU,
    double *Layer3_Gamma_GPU,
    double *Layer3_Beta_GPU
)
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 113 * 113)   // channel to work with
                        + (blockIdx.y * 32 * 113)    // Position in the grid row-wise
                        + (blockIdx.z * 32)          // Position in the grid column-wise
                        + (threadIdx.x * 113)
                        + (threadIdx.y);

    int weight_Position = filter_number * 32;

    int input_Position = (blockIdx.y * 32 * 112) // Position in the grid row-wise
                       + (blockIdx.z * 32)         // Position in the grid column-wise
                       + (threadIdx.x * 112)
                       + (threadIdx.y);

    for(int channel = 0; channel < 32; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer3_Neurons_GPU[(channel * 112 * 112) + input_Position] * Layer3_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer3_Mean_GPU[filter_number]) / Layer3_StanDev_GPU[filter_number];
    Z = (Z * Layer3_Gamma_GPU[filter_number]) + Layer3_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer4_Neurons_GPU[output_Position] = Z;
}

__global__ void executeThirdLayer_partB(double *Layer3_Neurons_GPU,
    double *Layer3_Weights_GPU,
    double *Layer4_Neurons_GPU,
    double *Layer3_Mean_GPU,
    double *Layer3_StanDev_GPU,
    double *Layer3_Gamma_GPU,
    double *Layer3_Beta_GPU
)
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 113 * 113)   // channel to work with
                        + (blockIdx.y * 16 * 113 + 96)  // Position in the grid row-wise and there is no column-wise position
                        + (threadIdx.x * 113)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 32;

    int input_Position = (blockIdx.y * 16 * 112)         // Position in the grid row-wise
                       + (96)                   // Position in the grid column-wise
                       + (threadIdx.x * 112)
                       + (threadIdx.y);

    for(int channel = 0 ; channel < 32 ; channel++) // Channel loop as we have 32 input channels to work with
    {
        product += (Layer3_Neurons_GPU[(channel * 112 * 112) + input_Position] * Layer3_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer3_Mean_GPU[filter_number]) / Layer3_StanDev_GPU[filter_number];
    Z = (Z * Layer3_Gamma_GPU[filter_number]) + Layer3_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer4_Neurons_GPU[output_Position] = Z;
}

__global__ void executeThirdLayer_partC(double *Layer3_Neurons_GPU,
    double *Layer3_Weights_GPU,
    double *Layer4_Neurons_GPU,
    double *Layer3_Mean_GPU,
    double *Layer3_StanDev_GPU,
    double *Layer3_Gamma_GPU,
    double *Layer3_Beta_GPU
)
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 113 * 113)   // channel to work with
                        + (96 * 113)                    // Position in the grid row-wise as row is last
                        + (blockIdx.y * 16)             // Position in the grid column-wise
                        + (threadIdx.x * 113)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 32;

    int input_Position = (96 * 112)            // row-wise: the bottom part of the grid after 96th row
                       + (blockIdx.y * 16)     // column-wise: block number in the 6 blocks of 16 * 16 threads
                       + (threadIdx.x * 112)   // Position inside one the above block row-wise
                       + (threadIdx.y);        // Position inside one the above block column-wise
    
    for(int channel = 0 ; channel < 32 ; channel++) // Channel loop as we have 32 input channels to work with
    {
        product += (Layer3_Neurons_GPU[(channel * 112 * 112) + input_Position] * Layer3_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer3_Mean_GPU[filter_number]) / Layer3_StanDev_GPU[filter_number];
    Z = (Z * Layer3_Gamma_GPU[filter_number]) + Layer3_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer4_Neurons_GPU[output_Position] = Z;
}
/*  ************************************************** THIRD LAYER END ********************************************************* */

/*  ************************************************** FOURTH LAYER START ****************************************************** */
__global__ void executeFourthLayer_partA(double *Layer4_Neurons_GPU,
    double *Layer4_Weights_GPU,
    double *Layer5_Neurons_GPU,
    double *Layer4_Mean_GPU,
    double *Layer4_StanDev_GPU,
    double *Layer4_Gamma_GPU,
    double *Layer4_Beta_GPU
)
{
    double product = 0.0;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (threadIdx.x * 113 * stride )
                       + (threadIdx.y * stride);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113)] * Layer4_Weights_GPU[weight_Position + (row * 3)])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 1] * Layer4_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 2] * Layer4_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer4_Mean_GPU[filter_number]) / Layer4_StanDev_GPU[filter_number];
    Z = (Z * Layer4_Gamma_GPU[filter_number]) + Layer4_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer5_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFourthLayer_partB(double *Layer4_Neurons_GPU,
    double *Layer4_Weights_GPU,
    double *Layer5_Neurons_GPU,
    double *Layer4_Mean_GPU,
    double *Layer4_StanDev_GPU,
    double *Layer4_Gamma_GPU,
    double *Layer4_Beta_GPU
)
{
    double product = 0.0;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (threadIdx.x * 56)
                        + (threadIdx.y + 32);

    int weight_Position = filter_number * 9;

    int input_Position = (threadIdx.x * 113 * stride)
                       + (threadIdx.y * stride) 
                       + (32 * stride);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113)] * Layer4_Weights_GPU[weight_Position + (row * 3)])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 1] * Layer4_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 2] * Layer4_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer4_Mean_GPU[filter_number]) / Layer4_StanDev_GPU[filter_number];
    Z = (Z * Layer4_Gamma_GPU[filter_number]) + Layer4_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer5_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFourthLayer_partC(double *Layer4_Neurons_GPU,
    double *Layer4_Weights_GPU,
    double *Layer5_Neurons_GPU,
    double *Layer4_Mean_GPU,
    double *Layer4_StanDev_GPU,
    double *Layer4_Gamma_GPU,
    double *Layer4_Beta_GPU
)
{
    double product = 0.0;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (56 * 32)
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (113 * 32 * stride)
                       + (threadIdx.x * 113 * stride)
                       + (threadIdx.y * stride);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113)] * Layer4_Weights_GPU[weight_Position + (row * 3)])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 1] * Layer4_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 2] * Layer4_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer4_Mean_GPU[filter_number]) / Layer4_StanDev_GPU[filter_number];
    Z = (Z * Layer4_Gamma_GPU[filter_number]) + Layer4_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer5_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFourthLayer_partD(double *Layer4_Neurons_GPU,
    double *Layer4_Weights_GPU,
    double *Layer5_Neurons_GPU,
    double *Layer4_Mean_GPU,
    double *Layer4_StanDev_GPU,
    double *Layer4_Gamma_GPU,
    double *Layer4_Beta_GPU
)
{
    double product = 0.0;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (56 * 32) 
                        + 32
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (113 * 32 * stride)
                       + (32 * stride)
                       + (threadIdx.x * 113 * stride)
                       + (threadIdx.y * stride);

    for(int row = 0; row < 3; row++) // This is the Row Loop
    {
        product += ((Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113)] * Layer4_Weights_GPU[weight_Position + (row * 3)])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 1] * Layer4_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 2] * Layer4_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer4_Mean_GPU[filter_number]) / Layer4_StanDev_GPU[filter_number];
    Z = (Z * Layer4_Gamma_GPU[filter_number]) + Layer4_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer5_Neurons_GPU[output_Position] = Z;
}


/*  ************************************************** FOURTH LAYER END ****************************************************** */

/*  *************************************************** FIFTH LAYER START **************************************************** */

__global__ void executeFifthLayer_partA(double *Layer5_Neurons_GPU,
    double *Layer5_Weights_GPU,
    double *Layer6_Neurons_GPU,
    double *Layer5_Mean_GPU,
    double *Layer5_StanDev_GPU,
    double *Layer5_Gamma_GPU,
    double *Layer5_Beta_GPU
)
{
    double product = 0.0;
    int offset = 59;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 58 * 58)   // channel to work with
                        + (threadIdx.x * 58)
                        + (threadIdx.y);

    int weight_Position = filter_number * 64;

    int input_Position = (threadIdx.x * 56)
                       + (threadIdx.y);

    for(int channel = 0; channel < 64; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer5_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer5_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer5_Mean_GPU[filter_number]) / Layer5_StanDev_GPU[filter_number];
    Z = (Z * Layer5_Gamma_GPU[filter_number]) + Layer5_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer6_Neurons_GPU[output_Position + offset] = Z;
}

__global__ void executeFifthLayer_partB(double *Layer5_Neurons_GPU,
    double *Layer5_Weights_GPU,
    double *Layer6_Neurons_GPU,
    double *Layer5_Mean_GPU,
    double *Layer5_StanDev_GPU,
    double *Layer5_Gamma_GPU,
    double *Layer5_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;
    int offset = 59;

    // Output position
    int output_Position = (filter_number * 58 * 58)   // channel to work with
                        + (threadIdx.x * 58)
                        + (threadIdx.y + 32);

    int weight_Position = filter_number * 64;

    int input_Position = (threadIdx.x * 56)
                       + (threadIdx.y) 
                       + (32);

    for(int channel = 0; channel < 64; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer5_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer5_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer5_Mean_GPU[filter_number]) / Layer5_StanDev_GPU[filter_number];
    Z = (Z * Layer5_Gamma_GPU[filter_number]) + Layer5_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer6_Neurons_GPU[output_Position + offset] = Z;
}

__global__ void executeFifthLayer_partC(double *Layer5_Neurons_GPU,
    double *Layer5_Weights_GPU,
    double *Layer6_Neurons_GPU,
    double *Layer5_Mean_GPU,
    double *Layer5_StanDev_GPU,
    double *Layer5_Gamma_GPU,
    double *Layer5_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;
    int offset = 59;

    // Output position
    int output_Position = (filter_number * 58 * 58)   // channel to work with
                        + (58 * 32)
                        + (threadIdx.x * 58)
                        + (threadIdx.y);

    int weight_Position = filter_number * 64;

    int input_Position = (56 * 32)
                       + (threadIdx.x * 56)
                       + (threadIdx.y);

    for(int channel = 0; channel < 64; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer5_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer5_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer5_Mean_GPU[filter_number]) / Layer5_StanDev_GPU[filter_number];
    Z = (Z * Layer5_Gamma_GPU[filter_number]) + Layer5_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer6_Neurons_GPU[output_Position + offset] = Z;
}

__global__ void executeFifthLayer_partD(double *Layer5_Neurons_GPU,
    double *Layer5_Weights_GPU,
    double *Layer6_Neurons_GPU,
    double *Layer5_Mean_GPU,
    double *Layer5_StanDev_GPU,
    double *Layer5_Gamma_GPU,
    double *Layer5_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;
    int offset = 59;

    // Output position
    int output_Position = (filter_number * 58 * 58)   // channel to work with
                        + (58 * 32) 
                        + 32
                        + (threadIdx.x * 58)
                        + (threadIdx.y);

    int weight_Position = filter_number * 64;

    int input_Position = (56 * 32)
                       + (32)
                       + (threadIdx.x * 56)
                       + (threadIdx.y);

    for(int channel = 0; channel < 64; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer5_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer5_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer5_Mean_GPU[filter_number]) / Layer5_StanDev_GPU[filter_number];
    Z = (Z * Layer5_Gamma_GPU[filter_number]) + Layer5_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer6_Neurons_GPU[output_Position + offset] = Z;
}

/*  *************************************************** FIFTH LAYER END **************************************************** */

/*  *************************************************** SIXTH LAYER START ************************************************** */
__global__ void executeSixthLayer_partA(double *Layer6_Neurons_GPU,
    double *Layer6_Weights_GPU,
    double *Layer7_Neurons_GPU,
    double *Layer6_Mean_GPU,
    double *Layer6_StanDev_GPU,
    double *Layer6_Gamma_GPU,
    double *Layer6_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (threadIdx.x * 58)
                       + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58)] * Layer6_Weights_GPU[weight_Position + (row * 3)])
                + (Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58) + 1] * Layer6_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58) + 2] * Layer6_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer6_Mean_GPU[filter_number]) / Layer6_StanDev_GPU[filter_number];
    Z = (Z * Layer6_Gamma_GPU[filter_number]) + Layer6_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer7_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSixthLayer_partB(double *Layer6_Neurons_GPU,
    double *Layer6_Weights_GPU,
    double *Layer7_Neurons_GPU,
    double *Layer6_Mean_GPU,
    double *Layer6_StanDev_GPU,
    double *Layer6_Gamma_GPU,
    double *Layer6_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (threadIdx.x * 56)
                        + (threadIdx.y + 32);

    int weight_Position = filter_number * 9;

    int input_Position = (threadIdx.x * 58)
                       + (threadIdx.y) 
                       + (32);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58)] * Layer6_Weights_GPU[weight_Position + (row * 3)])
                + (Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58) + 1] * Layer6_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58) + 2] * Layer6_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer6_Mean_GPU[filter_number]) / Layer6_StanDev_GPU[filter_number];
    Z = (Z * Layer6_Gamma_GPU[filter_number]) + Layer6_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer7_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSixthLayer_partC(double *Layer6_Neurons_GPU,
    double *Layer6_Weights_GPU,
    double *Layer7_Neurons_GPU,
    double *Layer6_Mean_GPU,
    double *Layer6_StanDev_GPU,
    double *Layer6_Gamma_GPU,
    double *Layer6_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (56 * 32)
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (58 * 32)
                       + (threadIdx.x * 58)
                       + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58)] * Layer6_Weights_GPU[weight_Position + (row * 3)])
                + (Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58) + 1] * Layer6_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58) + 2] * Layer6_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer6_Mean_GPU[filter_number]) / Layer6_StanDev_GPU[filter_number];
    Z = (Z * Layer6_Gamma_GPU[filter_number]) + Layer6_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer7_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSixthLayer_partD(double *Layer6_Neurons_GPU,
    double *Layer6_Weights_GPU,
    double *Layer7_Neurons_GPU,
    double *Layer6_Mean_GPU,
    double *Layer6_StanDev_GPU,
    double *Layer6_Gamma_GPU,
    double *Layer6_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (56 * 32) 
                        + 32
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (58 * 32)
                       + (32)
                       + (threadIdx.x * 58)
                       + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58)] * Layer6_Weights_GPU[weight_Position + (row * 3)])
                + (Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58) + 1] * Layer6_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer6_Neurons_GPU[(filter_number * 58 * 58) + input_Position + (row * 58) + 2] * Layer6_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer6_Mean_GPU[filter_number]) / Layer6_StanDev_GPU[filter_number];
    Z = (Z * Layer6_Gamma_GPU[filter_number]) + Layer6_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer7_Neurons_GPU[output_Position] = Z;
}

/*  *************************************************** SIXTH LAYER END **************************************************** */

/*  *************************************************** SEVENTH LAYER START ************************************************ */

__global__ void executeSeventhLayer_partA(double *Layer7_Neurons_GPU,
    double *Layer7_Weights_GPU,
    double *Layer8_Neurons_GPU,
    double *Layer7_Mean_GPU,
    double *Layer7_StanDev_GPU,
    double *Layer7_Gamma_GPU,
    double *Layer7_Beta_GPU
)
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 57 * 57)   // channel to work with
                        + (threadIdx.x * 57)
                        + (threadIdx.y);

    int weight_Position = filter_number * 128;

    int input_Position = (threadIdx.x * 56)
                       + (threadIdx.y);

    for(int channel = 0; channel < 128; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer7_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer7_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer7_Mean_GPU[filter_number]) / Layer7_StanDev_GPU[filter_number];
    Z = (Z * Layer7_Gamma_GPU[filter_number]) + Layer7_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer8_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSeventhLayer_partB(double *Layer7_Neurons_GPU,
    double *Layer7_Weights_GPU,
    double *Layer8_Neurons_GPU,
    double *Layer7_Mean_GPU,
    double *Layer7_StanDev_GPU,
    double *Layer7_Gamma_GPU,
    double *Layer7_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 57 * 57)   // channel to work with
                        + (threadIdx.x * 57)
                        + (threadIdx.y + 32);

    int weight_Position = filter_number * 128;

    int input_Position = (threadIdx.x * 56)
                       + (threadIdx.y) 
                       + (32);

    for(int channel = 0; channel < 128 ; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer7_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer7_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer7_Mean_GPU[filter_number]) / Layer7_StanDev_GPU[filter_number];
    Z = (Z * Layer7_Gamma_GPU[filter_number]) + Layer7_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer8_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSeventhLayer_partC(double *Layer7_Neurons_GPU,
    double *Layer7_Weights_GPU,
    double *Layer8_Neurons_GPU,
    double *Layer7_Mean_GPU,
    double *Layer7_StanDev_GPU,
    double *Layer7_Gamma_GPU,
    double *Layer7_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 57 * 57)   // channel to work with
                        + (57 * 32)
                        + (threadIdx.x * 57)
                        + (threadIdx.y);

    int weight_Position = filter_number * 128;

    int input_Position = (56 * 32)
                       + (threadIdx.x * 56)
                       + (threadIdx.y);

    for(int channel = 0; channel < 128 ; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer7_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer7_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer7_Mean_GPU[filter_number]) / Layer7_StanDev_GPU[filter_number];
    Z = (Z * Layer7_Gamma_GPU[filter_number]) + Layer7_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer8_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSeventhLayer_partD(double *Layer7_Neurons_GPU,
    double *Layer7_Weights_GPU,
    double *Layer8_Neurons_GPU,
    double *Layer7_Mean_GPU,
    double *Layer7_StanDev_GPU,
    double *Layer7_Gamma_GPU,
    double *Layer7_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 57 * 57)   // channel to work with
                        + (57 * 32) 
                        + 32
                        + (threadIdx.x * 57)
                        + (threadIdx.y);

    int weight_Position = filter_number * 128;

    int input_Position = (56 * 32)
                       + (32)
                       + (threadIdx.x * 56)
                       + (threadIdx.y);

    for(int channel = 0; channel < 128 ; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer7_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer7_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer7_Mean_GPU[filter_number]) / Layer7_StanDev_GPU[filter_number];
    Z = (Z * Layer7_Gamma_GPU[filter_number]) + Layer7_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer8_Neurons_GPU[output_Position] = Z;
}

/*  *************************************************** SEVENTH LAYER END **************************************************** */

/*  *************************************************** EIGHTH LAYER START ************************************************** */
__global__ void executeEighthLayer(double *Layer8_Neurons_GPU,
    double *Layer8_Weights_GPU,
    double *Layer9_Neurons_GPU,
    double *Layer8_Mean_GPU,
    double *Layer8_StanDev_GPU,
    double *Layer8_Gamma_GPU,
    double *Layer8_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;
    int stride = 2;

    // Output position
    int output_Position = (filter_number * 28 * 28)   // channel to work with
                        + (threadIdx.x * 28)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (threadIdx.x * 57 * stride)
                       + (threadIdx.y * stride);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer8_Neurons_GPU[(filter_number * 57 * 57) + input_Position + (row * 57)] * Layer8_Weights_GPU[weight_Position + (row * 3)])
                + (Layer8_Neurons_GPU[(filter_number * 57 * 57) + input_Position + (row * 57) + 1] * Layer8_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer8_Neurons_GPU[(filter_number * 57 * 57) + input_Position + (row * 57) + 2] * Layer8_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer8_Mean_GPU[filter_number]) / Layer8_StanDev_GPU[filter_number];
    Z = (Z * Layer8_Gamma_GPU[filter_number]) + Layer8_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer9_Neurons_GPU[output_Position] = Z;
}

/*  *************************************************** EIGHTH LAYER END **************************************************** */

/*  *************************************************** NINTH LAYER START ************************************************** */
__global__ void executeNinthLayer(double *Layer9_Neurons_GPU,
    double *Layer9_Weights_GPU,
    double *Layer10_Neurons_GPU,
    double *Layer9_Mean_GPU,
    double *Layer9_StanDev_GPU,
    double *Layer9_Gamma_GPU,
    double *Layer9_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;
    int offset = 31;

    // Output position
    int output_Position = (filter_number * 30 * 30)   // channel to work with
                        + (threadIdx.x * 30)
                        + (threadIdx.y);

    int weight_Position = filter_number * 128;

    int input_Position = (threadIdx.x * 28)
                        + (threadIdx.y);

    for(int channel = 0; channel < 128; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer9_Neurons_GPU[(channel * 28 * 28) + input_Position] * Layer9_Weights_GPU[weight_Position + channel]);
    }         

    double Z = (product - Layer9_Mean_GPU[filter_number]) / Layer9_StanDev_GPU[filter_number];
    Z = (Z * Layer9_Gamma_GPU[filter_number]) + Layer9_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer10_Neurons_GPU[output_Position + offset] = Z;
}

/*  *************************************************** NINTH LAYER END **************************************************** */

/*  *************************************************** TENTH LAYER START ************************************************** */
__global__ void executeTenthLayer(double *Layer10_Neurons_GPU,
    double *Layer10_Weights_GPU,
    double *Layer11_Neurons_GPU,
    double *Layer10_Mean_GPU,
    double *Layer10_StanDev_GPU,
    double *Layer10_Gamma_GPU,
    double *Layer10_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 28 * 28)   // channel to work with
                        + (threadIdx.x * 28)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (threadIdx.x * 30)
                       + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer10_Neurons_GPU[(filter_number * 30 * 30) + input_Position + (row * 30)] * Layer10_Weights_GPU[weight_Position + (row * 3)])
                + (Layer10_Neurons_GPU[(filter_number * 30 * 30) + input_Position + (row * 30) + 1] * Layer10_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer10_Neurons_GPU[(filter_number * 30 * 30) + input_Position + (row * 30) + 2] * Layer10_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer10_Mean_GPU[filter_number]) / Layer10_StanDev_GPU[filter_number];
    Z = (Z * Layer10_Gamma_GPU[filter_number]) + Layer10_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer11_Neurons_GPU[output_Position] = Z;
}

/*  *************************************************** TENTH LAYER END **************************************************** */

/*  *************************************************** ELEVENTH LAYER START ************************************************** */
__global__ void executeEleventhLayer(double *Layer11_Neurons_GPU,
    double *Layer11_Weights_GPU,
    double *Layer12_Neurons_GPU,
    double *Layer11_Mean_GPU,
    double *Layer11_StanDev_GPU,
    double *Layer11_Gamma_GPU,
    double *Layer11_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 29 * 29)   // channel to work with
                        + (threadIdx.x * 29)
                        + (threadIdx.y);

    int weight_Position = filter_number * 256;

    int input_Position = (threadIdx.x * 28)
                        + (threadIdx.y);

    for(int channel = 0; channel < 256; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer11_Neurons_GPU[(channel * 28 * 28) + input_Position] * Layer11_Weights_GPU[weight_Position + channel]);
    }         

    double Z = (product - Layer11_Mean_GPU[filter_number]) / Layer11_StanDev_GPU[filter_number];
    Z = (Z * Layer11_Gamma_GPU[filter_number]) + Layer11_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer12_Neurons_GPU[output_Position] = Z;
}

/*  *************************************************** ELEVENTH LAYER END **************************************************** */

/*  *************************************************** TWELFTH LAYER START ************************************************** */
__global__ void executeTwelfthLayer(double *Layer12_Neurons_GPU,
    double *Layer12_Weights_GPU,
    double *Layer13_Neurons_GPU,
    double *Layer12_Mean_GPU,
    double *Layer12_StanDev_GPU,
    double *Layer12_Gamma_GPU,
    double *Layer12_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;
    int stride = 2;

    // Output position
    int output_Position = (filter_number * 14 * 14)   // channel to work with
                        + (threadIdx.x * 14)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (threadIdx.x * 29 * stride)
                       + (threadIdx.y * stride);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer12_Neurons_GPU[(filter_number * 29 * 29) + input_Position + (row * 29)] * Layer12_Weights_GPU[weight_Position + (row * 3)])
                + (Layer12_Neurons_GPU[(filter_number * 29 * 29) + input_Position + (row * 29) + 1] * Layer12_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer12_Neurons_GPU[(filter_number * 29 * 29) + input_Position + (row * 29) + 2] * Layer12_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer12_Mean_GPU[filter_number]) / Layer12_StanDev_GPU[filter_number];
    Z = (Z * Layer12_Gamma_GPU[filter_number]) + Layer12_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer13_Neurons_GPU[output_Position] = Z;
}

/*  *************************************************** TWELFTH LAYER END **************************************************** */

