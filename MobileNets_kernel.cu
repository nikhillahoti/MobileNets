
#include <hip/hip_runtime.h>
#include <stdio.h>

/*  ************************************************** FIRST LAYER START ********************************************************* */
__global__ void executeFirstLayer_partA(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;
    int outputOffset = 115;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (blockIdx.y * 32 * 114)    // Position in the grid row-wise
                        + (blockIdx.z * 32)          // Position in the grid column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((blockIdx.y * 32 * 225) * stride) // Position in the grid row-wise
                       + (blockIdx.z * 32 * stride)         // Position in the grid column-wise
                       + (threadIdx.x * 225 * stride )
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer2_Neurons_GPU[output_Position + outputOffset] = Z;
}

__global__ void executeFirstLayer_partB(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;
    int outputOffset = 115;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (blockIdx.y * 16 * 114 + 96)  // Position in the grid row-wise and there is no column-wise position
                        + (threadIdx.x * 114)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((blockIdx.y * 16 * 225) * stride) + (96 * stride) // Position in the grid row-wise and column-wise
                       + (threadIdx.x * 225 * stride)
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer2_Neurons_GPU[output_Position + outputOffset] = Z;
}

__global__ void executeFirstLayer_partC(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;
    int outputOffset = 115;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (96 * 114)                    // Position in the grid row-wise as row is last
                        + (blockIdx.y * 16)             // Position in the grid column-wise
                        + (threadIdx.x * 114)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((96 * 225) * stride)
                       + (blockIdx.y * 16 * stride)     // Position in the grid row-wise and column-wise
                       + (threadIdx.x * 225 * stride)
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer2_Neurons_GPU[output_Position + outputOffset] = Z;
}
/*  ************************************************** FIRST LAYER END ************************************************************ */

/*  ************************************************** SECOND LAYER START ********************************************************* */
__global__ void executeSecondLayer_partA(double *Layer2_Neurons_GPU,
                            double *Layer2_Weights_GPU,
                            double *Layer3_Neurons_GPU,
                            double *Layer2_Mean_GPU,
                            double *Layer2_StanDev_GPU,
                            double *Layer2_Gamma_GPU,
                            double *Layer2_Beta_GPU
                        )
{
	double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 112 * 112)   // channel to work with
                        + (blockIdx.y * 32 * 112)    // Position in the grid row-wise
                        + (blockIdx.z * 32)          // Position in the grid column-wise
                        + (threadIdx.x * 112)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (blockIdx.y * 32 * 114) // Position in the grid row-wise
                       + (blockIdx.z * 32)         // Position in the grid column-wise
                       + (threadIdx.x * 114)
                       + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114)] * Layer2_Weights_GPU[weight_Position + (row * 3)])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 1] * Layer2_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 2] * Layer2_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer2_Mean_GPU[filter_number]) / Layer2_StanDev_GPU[filter_number];
    Z = (Z * Layer2_Gamma_GPU[filter_number]) + Layer2_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer3_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSecondLayer_partB(double *Layer2_Neurons_GPU,
                                    double *Layer2_Weights_GPU,
                                    double *Layer3_Neurons_GPU,
                                    double *Layer2_Mean_GPU,
                                    double *Layer2_StanDev_GPU,
                                    double *Layer2_Gamma_GPU,
                                    double *Layer2_Beta_GPU
                                )
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 112 * 112)   // channel to work with
                        + (blockIdx.y * 16 * 112 + 96)  // Position in the grid row-wise and there is no column-wise position
                        + (threadIdx.x * 112)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position  = (blockIdx.y * 16 * 114) 
                        + (96) // Position in the grid row-wise and column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114)] * Layer2_Weights_GPU[weight_Position + (row * 3)])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 1] * Layer2_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 2] * Layer2_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer2_Mean_GPU[filter_number]) / Layer2_StanDev_GPU[filter_number];
    Z = (Z * Layer2_Gamma_GPU[filter_number]) + Layer2_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 
    
    Layer3_Neurons_GPU[output_Position] = Z;
}

__global__ void executeSecondLayer_partC(double *Layer2_Neurons_GPU,
                                    double *Layer2_Weights_GPU,
                                    double *Layer3_Neurons_GPU,
                                    double *Layer2_Mean_GPU,
                                    double *Layer2_StanDev_GPU,
                                    double *Layer2_Gamma_GPU,
                                    double *Layer2_Beta_GPU
                                )
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 112 * 112)   // channel to work with
                        + (96 * 112)                    // Position in the grid row-wise as row is last
                        + (blockIdx.y * 16)             // Position in the grid column-wise
                        + (threadIdx.x * 112)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (96 * 114)
                        + (blockIdx.y * 16)     // Position in the grid row-wise and column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114)] * Layer2_Weights_GPU[weight_Position + (row * 3)])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 1] * Layer2_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 2] * Layer2_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer2_Mean_GPU[filter_number]) / Layer2_StanDev_GPU[filter_number];
    Z = (Z * Layer2_Gamma_GPU[filter_number]) + Layer2_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer3_Neurons_GPU[output_Position] = Z;
}
/*  ************************************************** SECOND LAYER END ********************************************************* */

/*  ************************************************** THIRD LAYER START ******************************************************** */
__global__ void executeThirdLayer_partA(double *Layer3_Neurons_GPU,
    double *Layer3_Weights_GPU,
    double *Layer4_Neurons_GPU,
    double *Layer3_Mean_GPU,
    double *Layer3_StanDev_GPU,
    double *Layer3_Gamma_GPU,
    double *Layer3_Beta_GPU
)
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 113 * 113)   // channel to work with
                        + (blockIdx.y * 32 * 113)    // Position in the grid row-wise
                        + (blockIdx.z * 32)          // Position in the grid column-wise
                        + (threadIdx.x * 113)
                        + (threadIdx.y);

    int weight_Position = filter_number * 32;

    int input_Position = (blockIdx.y * 32 * 112) // Position in the grid row-wise
                       + (blockIdx.z * 32)         // Position in the grid column-wise
                       + (threadIdx.x * 112)
                       + (threadIdx.y);

    for(int channel = 0; channel < 32; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer3_Neurons_GPU[(channel * 112 * 112) + input_Position] * Layer3_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer3_Mean_GPU[filter_number]) / Layer3_StanDev_GPU[filter_number];
    Z = (Z * Layer3_Gamma_GPU[filter_number]) + Layer3_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer4_Neurons_GPU[output_Position] = Z;
}

__global__ void executeThirdLayer_partB(double *Layer3_Neurons_GPU,
    double *Layer3_Weights_GPU,
    double *Layer4_Neurons_GPU,
    double *Layer3_Mean_GPU,
    double *Layer3_StanDev_GPU,
    double *Layer3_Gamma_GPU,
    double *Layer3_Beta_GPU
)
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 113 * 113)   // channel to work with
                        + (blockIdx.y * 16 * 113 + 96)  // Position in the grid row-wise and there is no column-wise position
                        + (threadIdx.x * 113)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 32;

    int input_Position = (blockIdx.y * 16 * 112)         // Position in the grid row-wise
                       + (96)                   // Position in the grid column-wise
                       + (threadIdx.x * 112)
                       + (threadIdx.y);

    for(int channel = 0 ; channel < 32 ; channel++) // Channel loop as we have 32 input channels to work with
    {
        product += (Layer3_Neurons_GPU[(channel * 112 * 112) + input_Position] * Layer3_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer3_Mean_GPU[filter_number]) / Layer3_StanDev_GPU[filter_number];
    Z = (Z * Layer3_Gamma_GPU[filter_number]) + Layer3_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer4_Neurons_GPU[output_Position] = Z;
}

__global__ void executeThirdLayer_partC(double *Layer3_Neurons_GPU,
    double *Layer3_Weights_GPU,
    double *Layer4_Neurons_GPU,
    double *Layer3_Mean_GPU,
    double *Layer3_StanDev_GPU,
    double *Layer3_Gamma_GPU,
    double *Layer3_Beta_GPU
)
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 113 * 113)   // channel to work with
                        + (96 * 113)                    // Position in the grid row-wise as row is last
                        + (blockIdx.y * 16)             // Position in the grid column-wise
                        + (threadIdx.x * 113)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 32;

    int input_Position = (96 * 112)            // row-wise: the bottom part of the grid after 96th row
                       + (blockIdx.y * 16)     // column-wise: block number in the 6 blocks of 16 * 16 threads
                       + (threadIdx.x * 112)   // Position inside one the above block row-wise
                       + (threadIdx.y);        // Position inside one the above block column-wise
    
    for(int channel = 0 ; channel < 32 ; channel++) // Channel loop as we have 32 input channels to work with
    {
        product += (Layer3_Neurons_GPU[(channel * 112 * 112) + input_Position] * Layer3_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer3_Mean_GPU[filter_number]) / Layer3_StanDev_GPU[filter_number];
    Z = (Z * Layer3_Gamma_GPU[filter_number]) + Layer3_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer4_Neurons_GPU[output_Position] = Z;
}
/*  ************************************************** THIRD LAYER END ********************************************************* */

/*  ************************************************** FOURTH LAYER START ****************************************************** */
__global__ void executeFourthLayer_partA(double *Layer4_Neurons_GPU,
    double *Layer4_Weights_GPU,
    double *Layer5_Neurons_GPU,
    double *Layer4_Mean_GPU,
    double *Layer4_StanDev_GPU,
    double *Layer4_Gamma_GPU,
    double *Layer4_Beta_GPU
)
{
    double product = 0.0;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (threadIdx.x * 113 * stride )
                       + (threadIdx.y * stride);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113)] * Layer4_Weights_GPU[weight_Position + (row * 3)])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 1] * Layer4_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 2] * Layer4_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer4_Mean_GPU[filter_number]) / Layer4_StanDev_GPU[filter_number];
    Z = (Z * Layer4_Gamma_GPU[filter_number]) + Layer4_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer5_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFourthLayer_partB(double *Layer4_Neurons_GPU,
    double *Layer4_Weights_GPU,
    double *Layer5_Neurons_GPU,
    double *Layer4_Mean_GPU,
    double *Layer4_StanDev_GPU,
    double *Layer4_Gamma_GPU,
    double *Layer4_Beta_GPU
)
{
    double product = 0.0;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (threadIdx.x * 56)
                        + (threadIdx.y + 32);

    int weight_Position = filter_number * 9;

    int input_Position = (threadIdx.x * 113 * stride)
                       + (threadIdx.y * stride) 
                       + (32 * stride);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113)] * Layer4_Weights_GPU[weight_Position + (row * 3)])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 1] * Layer4_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 2] * Layer4_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer4_Mean_GPU[filter_number]) / Layer4_StanDev_GPU[filter_number];
    Z = (Z * Layer4_Gamma_GPU[filter_number]) + Layer4_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer5_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFourthLayer_partC(double *Layer4_Neurons_GPU,
    double *Layer4_Weights_GPU,
    double *Layer5_Neurons_GPU,
    double *Layer4_Mean_GPU,
    double *Layer4_StanDev_GPU,
    double *Layer4_Gamma_GPU,
    double *Layer4_Beta_GPU
)
{
    double product = 0.0;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (56 * 32)
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (113 * 32 * stride)
                       + (threadIdx.x * 113 * stride)
                       + (threadIdx.y * stride);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113)] * Layer4_Weights_GPU[weight_Position + (row * 3)])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 1] * Layer4_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 2] * Layer4_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer4_Mean_GPU[filter_number]) / Layer4_StanDev_GPU[filter_number];
    Z = (Z * Layer4_Gamma_GPU[filter_number]) + Layer4_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer5_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFourthLayer_partD(double *Layer4_Neurons_GPU,
    double *Layer4_Weights_GPU,
    double *Layer5_Neurons_GPU,
    double *Layer4_Mean_GPU,
    double *Layer4_StanDev_GPU,
    double *Layer4_Gamma_GPU,
    double *Layer4_Beta_GPU
)
{
    double product = 0.0;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (56 * 32) 
                        + 32
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (113 * 32 * stride)
                       + (32 * stride)
                       + (threadIdx.x * 113 * stride)
                       + (threadIdx.y * stride);

    for(int row = 0; row < 3; row++) // This is the Row Loop
    {
        product += ((Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113)] * Layer4_Weights_GPU[weight_Position + (row * 3)])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 1] * Layer4_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer4_Neurons_GPU[(filter_number * 113 * 113) + input_Position + (row * 113) + 2] * Layer4_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    double Z = (product - Layer4_Mean_GPU[filter_number]) / Layer4_StanDev_GPU[filter_number];
    Z = (Z * Layer4_Gamma_GPU[filter_number]) + Layer4_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer5_Neurons_GPU[output_Position] = Z;
}


/*  ************************************************** FOURTH LAYER END ****************************************************** */

/*  *************************************************** FIFTH LAYER START **************************************************** */

__global__ void executeFifthLayer_partA(double *Layer5_Neurons_GPU,
    double *Layer5_Weights_GPU,
    double *Layer6_Neurons_GPU,
    double *Layer5_Mean_GPU,
    double *Layer5_StanDev_GPU,
    double *Layer5_Gamma_GPU,
    double *Layer5_Beta_GPU
)
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 64;

    int input_Position = (threadIdx.x * 56)
                       + (threadIdx.y);

    for(int channel = 0; channel < 64; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer5_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer5_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer5_Mean_GPU[filter_number]) / Layer5_StanDev_GPU[filter_number];
    Z = (Z * Layer5_Gamma_GPU[filter_number]) + Layer5_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer6_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFifthLayer_partB(double *Layer5_Neurons_GPU,
    double *Layer5_Weights_GPU,
    double *Layer6_Neurons_GPU,
    double *Layer5_Mean_GPU,
    double *Layer5_StanDev_GPU,
    double *Layer5_Gamma_GPU,
    double *Layer5_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (threadIdx.x * 56)
                        + (threadIdx.y + 32);

    int weight_Position = filter_number * 64;

    int input_Position = (threadIdx.x * 56)
                       + (threadIdx.y) 
                       + (32);

    for(int channel = 0; channel < 64; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer5_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer5_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer5_Mean_GPU[filter_number]) / Layer5_StanDev_GPU[filter_number];
    Z = (Z * Layer5_Gamma_GPU[filter_number]) + Layer5_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer6_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFifthLayer_partC(double *Layer5_Neurons_GPU,
    double *Layer5_Weights_GPU,
    double *Layer6_Neurons_GPU,
    double *Layer5_Mean_GPU,
    double *Layer5_StanDev_GPU,
    double *Layer5_Gamma_GPU,
    double *Layer5_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (56 * 32)
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 64;

    int input_Position = (56 * 32)
                       + (threadIdx.x * 56)
                       + (threadIdx.y);

    for(int channel = 0; channel < 64; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer5_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer5_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer5_Mean_GPU[filter_number]) / Layer5_StanDev_GPU[filter_number];
    Z = (Z * Layer5_Gamma_GPU[filter_number]) + Layer5_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer6_Neurons_GPU[output_Position] = Z;
}

__global__ void executeFifthLayer_partD(double *Layer5_Neurons_GPU,
    double *Layer5_Weights_GPU,
    double *Layer6_Neurons_GPU,
    double *Layer5_Mean_GPU,
    double *Layer5_StanDev_GPU,
    double *Layer5_Gamma_GPU,
    double *Layer5_Beta_GPU
)
{
    double product = 0.0;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 56 * 56)   // channel to work with
                        + (56 * 32) 
                        + 32
                        + (threadIdx.x * 56)
                        + (threadIdx.y);

    int weight_Position = filter_number * 64;

    int input_Position = (56 * 32)
                       + (32)
                       + (threadIdx.x * 56)
                       + (threadIdx.y);

    for(int channel = 0; channel < 64; channel++)       // This is the channel loop as we have 32 channels to work with
    {
        product += (Layer5_Neurons_GPU[(channel * 56 * 56) + input_Position] * Layer5_Weights_GPU[weight_Position + channel]);
    }

    double Z = (product - Layer5_Mean_GPU[filter_number]) / Layer5_StanDev_GPU[filter_number];
    Z = (Z * Layer5_Gamma_GPU[filter_number]) + Layer5_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer6_Neurons_GPU[output_Position] = Z;
}

/*  *************************************************** FIFTH LAYER END **************************************************** */