
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void executeFirstLayer_partA(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;
    int outputOffset = 115;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (blockIdx.y * 32 * 114)    // Position in the grid row-wise
                        + (blockIdx.z * 32)          // Position in the grid column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((blockIdx.y * 32 * 225) * stride) // Position in the grid row-wise
                       + (blockIdx.z * 32 * stride)         // Position in the grid column-wise
                       + (threadIdx.x * 225 * stride )
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer2_Neurons_GPU[output_Position + outputOffset] = Z;
}


__global__ void executeFirstLayer_partB(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;
    int outputOffset = 115;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (blockIdx.y * 16 * 114 + 96)  // Position in the grid row-wise and there is no column-wise position
                        + (threadIdx.x * 114)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((blockIdx.y * 16 * 225) * stride) + (96 * stride) // Position in the grid row-wise and column-wise
                       + (threadIdx.x * 225 * stride)
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer2_Neurons_GPU[output_Position + outputOffset] = Z;
}

__global__ void executeFirstLayer_partC(double *Layer1_Neurons_GPU,
                            double *Layer1_Weights_GPU,
                            double *Layer2_Neurons_GPU,
                            double *Layer1_Mean_GPU,
                            double *Layer1_StanDev_GPU,
                            double *Layer1_Gamma_GPU,
                            double *Layer1_Beta_GPU
                        )
{
	double product = 0.0;
    int outputOffset = 115;
    int stride = 2;
    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 114 * 114)   // channel to work with
                        + (96 * 114)                    // Position in the grid row-wise as row is last
                        + (blockIdx.y * 16)             // Position in the grid column-wise
                        + (threadIdx.x * 114)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 27;

    int input_Position = ((96 * 225) * stride)
                       + (blockIdx.y * 16 * stride)     // Position in the grid row-wise and column-wise
                       + (threadIdx.x * 225 * stride)
                       + (threadIdx.y * stride);

    /* RGB weights and input 3*3*3 */
    for(int channel = 0; channel < 3; channel++) // This is the Channel loop
    {
        for(int row = 0; row < 3; row++)       // This is the Row Loop
        {
            product += ((Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225)] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3)])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 1] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 1])
                    + (Layer1_Neurons_GPU[(channel * 225 * 225) + input_Position + (row * 225) + 2] * Layer1_Weights_GPU[weight_Position + (channel * 3 * 3) + (row * 3) + 2]));
        }
    }

    double Z = (product - Layer1_Mean_GPU[filter_number]) / Layer1_StanDev_GPU[filter_number];
    Z = (Z * Layer1_Gamma_GPU[filter_number]) + Layer1_Beta_GPU[filter_number];

    // ReLU Layer
    if(Z < 0)
        Z = 0; // max(0,x)

    // ReLU 6 Layer
    if(Z > 6)
        Z = 6.0; 

    Layer2_Neurons_GPU[output_Position + outputOffset] = Z;
}


// Second Layer
__global__ void executeSecondLayer_partA(double *Layer2_Neurons_GPU,
                            double *Layer2_Weights_GPU,
                            double *Layer3_Neurons_GPU
                        )
{
	double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 112 * 112)   // channel to work with
                        + (blockIdx.y * 32 * 112)    // Position in the grid row-wise
                        + (blockIdx.z * 32)          // Position in the grid column-wise
                        + (threadIdx.x * 112)
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (blockIdx.y * 32 * 114) // Position in the grid row-wise
                       + (blockIdx.z * 32)         // Position in the grid column-wise
                       + (threadIdx.x * 114)
                       + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114)] * Layer2_Weights_GPU[weight_Position + (row * 3)])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 1] * Layer2_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 2] * Layer2_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    Layer3_Neurons_GPU[output_Position] = product;
}

__global__ void executeSecondLayer_partB(double *Layer2_Neurons_GPU,
                                    double *Layer2_Weights_GPU,
                                    double *Layer3_Neurons_GPU
                                )
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 112 * 112)   // channel to work with
                        + (blockIdx.y * 16 * 112 + 96)  // Position in the grid row-wise and there is no column-wise position
                        + (threadIdx.x * 112)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position  = (blockIdx.y * 16 * 114) 
                        + (96) // Position in the grid row-wise and column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114)] * Layer2_Weights_GPU[weight_Position + (row * 3)])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 1] * Layer2_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 2] * Layer2_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    Layer3_Neurons_GPU[output_Position] = product;
}

__global__ void executeSecondLayer_partC(double *Layer2_Neurons_GPU,
                                    double *Layer2_Weights_GPU,
                                    double *Layer3_Neurons_GPU
                                )
{
    double product = 0.0;

    int filter_number = blockIdx.x;

    // Output position
    int output_Position = (filter_number * 112 * 112)   // channel to work with
                        + (96 * 112)                    // Position in the grid row-wise as row is last
                        + (blockIdx.y * 16)             // Position in the grid column-wise
                        + (threadIdx.x * 112)           // Position inside the 256 (16 * 16) block
                        + (threadIdx.y);

    int weight_Position = filter_number * 9;

    int input_Position = (96 * 114)
                        + (blockIdx.y * 16)     // Position in the grid row-wise and column-wise
                        + (threadIdx.x * 114)
                        + (threadIdx.y);

    for(int row = 0; row < 3; row++)       // This is the Row Loop
    {
        product += ((Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114)] * Layer2_Weights_GPU[weight_Position + (row * 3)])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 1] * Layer2_Weights_GPU[weight_Position + (row * 3) + 1])
                + (Layer2_Neurons_GPU[(filter_number * 114 * 114) + input_Position + (row * 114) + 2] * Layer2_Weights_GPU[weight_Position + (row * 3) + 2]));
    }

    Layer3_Neurons_GPU[output_Position] = product;
}

