#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>

#include "MobileNets_kernel.cu"

#define INPUT_LAYER_SIZE 225 * 225 * 3
#define FIRST_LAYER_WEIGHT_SIZE 32 * 3 * 3 * 3
#define FIRST_LAYER_OUTPUT_SIZE 112 * 112 * 32

// Function declarations 
void NeuralNetwork();
double * read_Input_Weights(char * weightFileName);
void read_Input_File(char * inputFileName, double *Layer1_Neurons_CPU);

int main(){
    NeuralNetwork();    
}

void NeuralNetwork(){
    // Reading the input layer data 
    double * Layer1_Neurons_CPU = (double *) malloc(sizeof(double) * INPUT_LAYER_SIZE);
    // read_Input_File("data/Input_Layer_Data.txt", Layer1_Neurons_CPU);
    read_Input_File("data/fTemp.txt", Layer1_Neurons_CPU);

    // Reading the weights file
    double * Layer1_Weights_CPU = read_Input_Weights("data/First_Layer_Weights.txt");

    // Allocating memory for Output Matrix
    double * Layer2_Neurons_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);

    // Copy memory from Host to Kernel
    double *Layer1_Weights_GPU, *Layer1_Neurons_GPU, *Layer2_Neurons_GPU;

    hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(double) * INPUT_LAYER_SIZE); 
    hipMalloc((void**) &Layer1_Weights_GPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);

    hipMemcpy(Layer1_Neurons_GPU, Layer1_Neurons_CPU, sizeof(double) * INPUT_LAYER_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Weights_GPU, Layer1_Weights_CPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Neurons_GPU, Layer2_Neurons_CPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE, hipMemcpyHostToDevice);

    // Kernel Launch 
    dim3 gridSizeA(32, 3, 3);
    dim3 blockSizeA(32,32);

    executeFirstLayer_partA<<< gridSizeA, blockSizeA>>>(Layer1_Neurons_GPU, Layer1_Weights_GPU, Layer2_Neurons_GPU);

    hipDeviceSynchronize();

    dim3 gridSizeB(32, 7);
    dim3 blockSizeB(16, 16);

    executeFirstLayer_partB<<< gridSizeB, blockSizeB>>>(Layer1_Neurons_GPU, Layer1_Weights_GPU, Layer2_Neurons_GPU);

    hipDeviceSynchronize();

    dim3 gridSizeC(32, 6);
    dim3 blockSizeC(16, 16);

    executeFirstLayer_partC<<< gridSizeC, blockSizeC>>>(Layer1_Neurons_GPU, Layer1_Weights_GPU, Layer2_Neurons_GPU);

    hipDeviceSynchronize();


    // Get back the data from the kernel to the host
    hipMemcpy(Layer2_Neurons_CPU, Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Logic to save into the file to verify the results
    FILE * fOutput = fopen("data/output.txt", "w");
    int value = FIRST_LAYER_OUTPUT_SIZE;
    for(int i = 0 ; i < value ; i++){
        fprintf (fOutput, "%lf\n", Layer2_Neurons_CPU[i]);
    }
    fclose(fOutput);

    printf("\n\n Processing Done !!! ");

    // Free the memory at the end
    free(Layer1_Neurons_CPU);
    free(Layer1_Weights_CPU);
    free(Layer2_Neurons_CPU);

    hipFree(Layer1_Neurons_GPU);
    hipFree(Layer1_Weights_GPU);
    hipFree(Layer2_Neurons_GPU);
}


double * read_Input_Weights(char * inputWeightFileName){
    
    // Allocate the memory 
    double * input_weight = (double *) malloc(sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);

    FILE *fp = fopen(inputWeightFileName, "r");
    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return NULL;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;

    while ((read = getline(&line, &len, fp)) != -1) 
        input_weight[counter++] = atof(line);
    fclose(fp);
    return input_weight;
}

void read_Input_File(char * inputFileName, double * Layer1_Neurons_CPU){
    FILE *fp = fopen(inputFileName, "r");

    int value = INPUT_LAYER_SIZE;
    printf("\n Value is %d", value);
    

    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;
    int index = 0;
    int lastRow = 0;

    while ((read = getline(&line, &len, fp)) != -1) {
        Layer1_Neurons_CPU[counter++] = atof(line);
        index++;
        // handle padding 
        if (index == 224){
            Layer1_Neurons_CPU[counter++] = 0;
            index = 0;
            lastRow++;
            if(lastRow == 224){
                lastRow = 0;
                int temp = 0;
                while (temp < 225) {
                    Layer1_Neurons_CPU[counter++] = 0;
                    temp++;
                }
            }
        }
    }

    printf("\n Total characters read ---> %d\n", counter);
    fclose(fp);
}

