#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include <unistd.h>

#include "MobileNets_kernel.cu"

#define INPUT_LAYER_SIZE 225 * 225 * 3
#define FIRST_LAYER_WEIGHT_SIZE 32 * 3 * 3 * 3
#define FIRST_LAYER_OUTPUT_SIZE 114 * 114 * 32
#define FIRST_LAYER_CHANNELS 32

#define SECOND_LAYER_WEIGHT_SIZE 32 * 3 * 3
#define SECOND_LAYER_OUTPUT_SIZE 112 * 112 * 32
#define SECOND_LAYER_CHANNELS 32

#define THIRD_LAYER_WEIGHT_SIZE 64 * 32
#define THIRD_LAYER_OUTPUT_SIZE 113 * 113 * 64
#define THIRD_LAYER_CHANNELS 64

#define FOURTH_LAYER_WEIGHT_SIZE 3 * 3 * 64
#define FOURTH_LAYER_OUTPUT_SIZE 56 * 56 * 64
#define FOURTH_LAYER_CHANNELS 64

#define FIFTH_LAYER_WEIGHT_SIZE 64 * 128
#define FIFTH_LAYER_OUTPUT_SIZE 58 * 58 * 128
#define FIFTH_LAYER_CHANNELS 128

#define SIXTH_LAYER_WEIGHT_SIZE 3 * 3 * 128
#define SIXTH_LAYER_OUTPUT_SIZE 56 * 56 * 128
#define SIXTH_LAYER_CHANNELS 128

#define SEVENTH_LAYER_WEIGHT_SIZE 128 * 128
#define SEVENTH_LAYER_OUTPUT_SIZE 57 * 57 * 128
#define SEVENTH_LAYER_CHANNELS 128

#define EIGHTH_LAYER_WEIGHT_SIZE 3 * 3 * 128
#define EIGHTH_LAYER_OUTPUT_SIZE 28 * 28 * 128
#define EIGHTH_LAYER_CHANNELS 128

#define NINTH_LAYER_WEIGHT_SIZE  128 * 256
#define NINTH_LAYER_OUTPUT_SIZE 30 * 30 * 256
#define NINTH_LAYER_CHANNELS 256

#define TENTH_LAYER_WEIGHT_SIZE  9 * 256
#define TENTH_LAYER_OUTPUT_SIZE 28 * 28 * 256
#define TENTH_LAYER_CHANNELS 256

#define ELEVENTH_LAYER_WEIGHT_SIZE  256 * 256
#define ELEVENTH_LAYER_OUTPUT_SIZE 29 * 29 * 256
#define ELEVENTH_LAYER_CHANNELS 256

#define TWELFTH_LAYER_WEIGHT_SIZE  9 * 256
#define TWELFTH_LAYER_OUTPUT_SIZE 14 * 14 * 256
#define TWELFTH_LAYER_CHANNELS 256

// Function declarations
void NeuralNetwork();
void read_File(const char * weightFileName, double *Layer1_Weights_CPU);
void read_Input_File(const char * inputFileName, double *Layer1_Neurons_CPU);

void Read_First_Layer_Data(double * Layer1_Neurons_CPU,
    double * Layer1_Weights_CPU,
    double * Layer1_Mean_CPU,
    double * Layer1_StanDev_CPU,
    double * Layer1_Gamma_CPU,
    double * Layer1_Beta_CPU
);

void Execute_First_Layer(double * Layer2_Neurons_GPU);

void Read_SecondLayer_Data(double *Layer1_Weights_CPU,
    double *Layer2_Mean_CPU,
    double *Layer2_StanDev_CPU,
    double *Layer2_Gamma_CPU,
    double *Layer2_Beta_CPU
);

void Execute_Second_Layer(
    double * Layer2_Neurons_GPU,
    double * Layer3_Neurons_GPU
);

void Read_ThirdLayer_Data(double *Layer3_Weights_CPU,
    double * Layer3_Mean_CPU,
    double * Layer3_StanDev_CPU,
    double * Layer3_Gamma_CPU,
    double * Layer3_Beta_CPU
);

void Execute_Third_Layer(
    double * Layer3_Neurons_GPU,
    double * Layer4_Neurons_GPU
);

void Read_FourthLayer_Data(double *Layer4_Weights_CPU,
    double * Layer4_Mean_CPU,
    double * Layer4_StanDev_CPU,
    double * Layer4_Gamma_CPU,
    double * Layer4_Beta_CPU
);

void Execute_Fourth_Layer(
    double * Layer4_Neurons_GPU,
    double * Layer5_Neurons_GPU
);

void Read_FifthLayer_Data(double *Layer5_Weights_CPU,
    double * Layer5_Mean_CPU,
    double * Layer5_StanDev_CPU,
    double * Layer5_Gamma_CPU,
    double * Layer5_Beta_CPU
);

void Execute_Fifth_Layer(
    double * Layer5_Neurons_GPU,
    double * Layer6_Neurons_GPU
);

void Read_SixthLayer_Data(double *Layer6_Weights_CPU,
    double * Layer6_Mean_CPU,
    double * Layer6_StanDev_CPU,
    double * Layer6_Gamma_CPU,
    double * Layer6_Beta_CPU
);

void Execute_Sixth_Layer(
    double * Layer6_Neurons_GPU,
    double * Layer7_Neurons_GPU
);

void Read_SeventhLayer_Data(double *Layer7_Weights_CPU,
    double * Layer7_Mean_CPU,
    double * Layer7_StanDev_CPU,
    double * Layer7_Gamma_CPU,
    double * Layer7_Beta_CPU
);

void Execute_Seventh_Layer(
    double * Layer7_Neurons_GPU,
    double * Layer8_Neurons_GPU
);

void Read_EighthLayer_Data(double *Layer8_Weights_CPU,
    double * Layer8_Mean_CPU,
    double * Layer8_StanDev_CPU,
    double * Layer8_Gamma_CPU,
    double * Layer8_Beta_CPU
);

void Execute_Eighth_Layer(
    double * Layer8_Neurons_GPU,
    double * Layer9_Neurons_GPU
);

void Read_NinthLayer_Data(double *Layer9_Weights_CPU,
    double * Layer9_Mean_CPU,
    double * Layer9_StanDev_CPU,
    double * Layer9_Gamma_CPU,
    double * Layer9_Beta_CPU
);

void Execute_Ninth_Layer(
    double * Layer9_Neurons_GPU,
    double * Layer10_Neurons_GPU
);

void Read_TenthLayer_Data(double *Layer10_Weights_CPU,
    double * Layer10_Mean_CPU,
    double * Layer10_StanDev_CPU,
    double * Layer10_Gamma_CPU,
    double * Layer10_Beta_CPU
);

void Execute_Tenth_Layer(
    double * Layer10_Neurons_GPU,
    double * Layer11_Neurons_GPU
);

void Read_EleventhLayer_Data(double *Layer11_Weights_CPU,
    double * Layer11_Mean_CPU,
    double * Layer11_StanDev_CPU,
    double * Layer11_Gamma_CPU,
    double * Layer11_Beta_CPU
);

void Execute_Eleventh_Layer(
    double * Layer11_Neurons_GPU,
    double * Layer12_Neurons_GPU
);

void Read_TwelvethLayer_Data(double *Layer12_Weights_CPU,
    double * Layer12_Mean_CPU,
    double * Layer12_StanDev_CPU,
    double * Layer12_Gamma_CPU,
    double * Layer12_Beta_CPU
);

void Execute_Twelveth_Layer(
    double * Layer12_Neurons_GPU,
    double * Layer13_Neurons_GPU
);

int main(){
    NeuralNetwork();
}

void NeuralNetwork(){
    FILE *fOutput;
    int value;

    /* ************************************************ FIRST LAYER ******************************************************** */
    double *Layer2_Neurons_GPU = NULL; 
    hipMalloc((void**) &Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);

    Execute_First_Layer(Layer2_Neurons_GPU);

    // Saving output of the first layer: Initially Not Saved
    bool SAVE_FIRST_LAYER_WEIGHTS = false;
    if(SAVE_FIRST_LAYER_WEIGHTS){
        
        double *Layer2_Neurons_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer2_Neurons_CPU, Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/FirstLayer/output.txt", "w");
        value = FIRST_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer2_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer2_Neurons_CPU);
    }
    
    printf("\n Layer 1 Execution complete !!!");
    /* ************************************************ FIRST LAYER COMPLETE *********************************************** */

    /* ************************************************ SECOND LAYER ******************************************************** */
    double *Layer3_Neurons_GPU;
    hipMalloc((void**) &Layer3_Neurons_GPU, sizeof(double) * SECOND_LAYER_OUTPUT_SIZE);

    Execute_Second_Layer(Layer2_Neurons_GPU, Layer3_Neurons_GPU);

    bool SAVE_SECOND_LAYER_WEIGHTS = false;
    if(SAVE_SECOND_LAYER_WEIGHTS){
        
        double * Layer3_Neurons_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer3_Neurons_CPU, Layer3_Neurons_GPU, sizeof(double) * SECOND_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/SecondLayer/output.txt", "w");
        value = SECOND_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer3_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer3_Neurons_CPU);
    }

    printf("\n Layer 2 Execution complete !!!");
    /* ************************************************ SECOND LAYER COMPLETE *********************************************** */

    /* ************************************************ THIRD LAYER ******************************************************** */
    double *Layer4_Neurons_GPU;
    hipMalloc((void**) &Layer4_Neurons_GPU, sizeof(double) * THIRD_LAYER_OUTPUT_SIZE);

    Execute_Third_Layer(Layer3_Neurons_GPU, Layer4_Neurons_GPU);

    bool SAVE_THIRD_LAYER_WEIGHTS = false;
    if(SAVE_THIRD_LAYER_WEIGHTS){
        double * Layer4_Neurons_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer4_Neurons_CPU, Layer4_Neurons_GPU, sizeof(double) * THIRD_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/ThirdLayer/output.txt", "w");
        value = THIRD_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer4_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer4_Neurons_CPU);
    }

    printf("\n Layer 3 Execution complete !!!");
    /* ************************************************ THIRD LAYER COMPLETE *********************************************** */

    /* ************************************************ FOURTH LAYER ******************************************************** */
    double *Layer5_Neurons_GPU;
    hipMalloc((void**) &Layer5_Neurons_GPU, sizeof(double) * FOURTH_LAYER_OUTPUT_SIZE);

    Execute_Fourth_Layer(Layer4_Neurons_GPU, Layer5_Neurons_GPU);

    bool SAVE_FOURTH_LAYER_WEIGHTS = false;
    if(SAVE_FOURTH_LAYER_WEIGHTS){
        double * Layer5_Neurons_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer5_Neurons_CPU, Layer5_Neurons_GPU, sizeof(double) * FOURTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/FourthLayer/output.txt", "w");
        value = FOURTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer5_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer5_Neurons_CPU);
    }

    printf("\n Layer 4 Execution complete !!!");
    /* ************************************************ FOURTH LAYER COMPLETE *********************************************** */

    /* ************************************************ FIFTH LAYER ******************************************************** */
    double *Layer6_Neurons_GPU;
    hipMalloc((void**) &Layer6_Neurons_GPU, sizeof(double) * FIFTH_LAYER_OUTPUT_SIZE);

    Execute_Fifth_Layer(Layer5_Neurons_GPU, Layer6_Neurons_GPU);

    bool SAVE_FIFTH_LAYER_WEIGHTS = false;
    if(SAVE_FIFTH_LAYER_WEIGHTS){
        double * Layer6_Neurons_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer6_Neurons_CPU, Layer6_Neurons_GPU, sizeof(double) * FIFTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/FifthLayer/output.txt", "w");
        value = FIFTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer6_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer6_Neurons_CPU);
    }

    printf("\n Layer 5 Execution complete !!!");
    /* ************************************************ FIFTH LAYER COMPLETE *********************************************** */

    /* ************************************************ SIXTH LAYER ******************************************************** */
    double *Layer7_Neurons_GPU;
    hipMalloc((void**) &Layer7_Neurons_GPU, sizeof(double) * SIXTH_LAYER_OUTPUT_SIZE);

    Execute_Sixth_Layer(Layer6_Neurons_GPU, Layer7_Neurons_GPU);

    bool SAVE_SIXTH_LAYER_WEIGHTS = false;
    if(SAVE_SIXTH_LAYER_WEIGHTS){
        double * Layer7_Neurons_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer7_Neurons_CPU, Layer7_Neurons_GPU, sizeof(double) * SIXTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/SixthLayer/output.txt", "w");
        value = SIXTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer7_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer7_Neurons_CPU);
    }

    printf("\n Layer 6 Execution complete !!!");
    /* ************************************************ SIXTH LAYER COMPLETE *********************************************** */

    /* ************************************************ SEVENTH LAYER START ******************************************************** */
    double *Layer8_Neurons_GPU;
    hipMalloc((void**) &Layer8_Neurons_GPU, sizeof(double) * SEVENTH_LAYER_OUTPUT_SIZE);

    Execute_Seventh_Layer(Layer7_Neurons_GPU, Layer8_Neurons_GPU);

    bool SAVE_SEVENTH_LAYER_WEIGHTS = false;
    if(SAVE_SEVENTH_LAYER_WEIGHTS){
        double * Layer8_Neurons_CPU = (double *) malloc(sizeof(double) * SEVENTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer8_Neurons_CPU, Layer8_Neurons_GPU, sizeof(double) * SEVENTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/SeventhLayer/output.txt", "w");
        value = SEVENTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer8_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer8_Neurons_CPU);
    }

    printf("\n Layer 7 Execution complete !!!");
    /* ************************************************ SEVENTH LAYER COMPLETE *********************************************** */

    /* ************************************************ EIGHTH LAYER START ******************************************************** */
    double *Layer9_Neurons_GPU;
    hipMalloc((void**) &Layer9_Neurons_GPU, sizeof(double) * EIGHTH_LAYER_OUTPUT_SIZE);

    Execute_Eighth_Layer(Layer8_Neurons_GPU, Layer9_Neurons_GPU);

    bool SAVE_EIGHTH_LAYER_WEIGHTS = false;
    if(SAVE_EIGHTH_LAYER_WEIGHTS){
        double * Layer9_Neurons_CPU = (double *) malloc(sizeof(double) * EIGHTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer9_Neurons_CPU, Layer9_Neurons_GPU, sizeof(double) * EIGHTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/EighthLayer/output.txt", "w");
        value = EIGHTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer9_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer9_Neurons_CPU);
    }

    printf("\n Layer 8 Execution complete !!!");
    /* ************************************************ EIGHTH LAYER COMPLETE *********************************************** */

    /* ************************************************ NINTH LAYER START ******************************************************** */
    double *Layer10_Neurons_GPU;
    hipMalloc((void**) &Layer10_Neurons_GPU, sizeof(double) * NINTH_LAYER_OUTPUT_SIZE);

    Execute_Ninth_Layer(Layer9_Neurons_GPU, Layer10_Neurons_GPU);

    bool SAVE_NINTH_LAYER_WEIGHTS = false;
    if(SAVE_NINTH_LAYER_WEIGHTS){
        double * Layer10_Neurons_CPU = (double *) malloc(sizeof(double) * NINTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer10_Neurons_CPU, Layer10_Neurons_GPU, sizeof(double) * NINTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/NinthLayer/output.txt", "w");
        value = NINTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer10_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer10_Neurons_CPU);
    }

    printf("\n Layer 9 Execution complete !!!");
    /* ************************************************ NINTH LAYER COMPLETE *********************************************** */

    /* ************************************************ TENTH LAYER START ******************************************************** */
    double *Layer11_Neurons_GPU;
    hipMalloc((void**) &Layer11_Neurons_GPU, sizeof(double) * TENTH_LAYER_OUTPUT_SIZE);

    Execute_Tenth_Layer(Layer10_Neurons_GPU, Layer11_Neurons_GPU);

    bool SAVE_TENTH_LAYER_WEIGHTS = false;
    if(SAVE_TENTH_LAYER_WEIGHTS){
        double * Layer11_Neurons_CPU = (double *) malloc(sizeof(double) * TENTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer11_Neurons_CPU, Layer11_Neurons_GPU, sizeof(double) * TENTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/TenthLayer/output.txt", "w");
        value = TENTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer11_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer11_Neurons_CPU);
    }

    printf("\n Layer 10 Execution complete !!!");
    /* ************************************************ TENTH LAYER COMPLETE *********************************************** */

    /* ************************************************ ELEVENTH LAYER START ******************************************************** */
    double *Layer12_Neurons_GPU;
    hipMalloc((void**) &Layer12_Neurons_GPU, sizeof(double) * ELEVENTH_LAYER_OUTPUT_SIZE);

    Execute_Eleventh_Layer(Layer11_Neurons_GPU, Layer12_Neurons_GPU);

    bool SAVE_ELEVENTH_LAYER_WEIGHTS = false;
    if(SAVE_ELEVENTH_LAYER_WEIGHTS){
        double * Layer12_Neurons_CPU = (double *) malloc(sizeof(double) * ELEVENTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer12_Neurons_CPU, Layer12_Neurons_GPU, sizeof(double) * ELEVENTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/EleventhLayer/output.txt", "w");
        value = ELEVENTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer12_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer12_Neurons_CPU);
    }

    printf("\n Layer 11 Execution complete !!!");
    /* ************************************************ ELEVENTH LAYER COMPLETE *********************************************** */

    // Deallocate Memory
    hipFree(Layer2_Neurons_GPU);
    hipFree(Layer3_Neurons_GPU);
    hipFree(Layer4_Neurons_GPU);
    hipFree(Layer5_Neurons_GPU);
    hipFree(Layer6_Neurons_GPU);
    hipFree(Layer7_Neurons_GPU);
    hipFree(Layer8_Neurons_GPU);
    hipFree(Layer9_Neurons_GPU);
    hipFree(Layer10_Neurons_GPU);
    hipFree(Layer11_Neurons_GPU);

    hipDeviceSynchronize();

    /* ************************************************ TWELVETH LAYER START ******************************************************** */
    double *Layer13_Neurons_GPU;
    hipMalloc((void**) &Layer13_Neurons_GPU, sizeof(double) * TWELFTH_LAYER_OUTPUT_SIZE);

    Execute_Twelveth_Layer(Layer12_Neurons_GPU, Layer13_Neurons_GPU);

    bool SAVE_TWELVETH_LAYER_WEIGHTS = true;
    if(SAVE_TWELVETH_LAYER_WEIGHTS){
        double * Layer13_Neurons_CPU = (double *) malloc(sizeof(double) * TWELFTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer13_Neurons_CPU, Layer13_Neurons_GPU, sizeof(double) * TWELFTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/TwelvethLayer/output.txt", "w");
        value = TWELFTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer13_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer13_Neurons_CPU);
    }

    printf("\n Layer 12 Execution complete !!!");
    /* ************************************************ TWELVETH LAYER COMPLETE *********************************************** */


    printf("\n\n Processing Done !!! \n\n");

    
    hipFree(Layer12_Neurons_GPU);
    //hipFree(Layer13_Neurons_GPU);
}

void Execute_First_Layer(double *Layer2_Neurons_GPU)
{
    double *Layer1_Neurons_CPU = (double *) malloc(sizeof(double) * INPUT_LAYER_SIZE);
    double *Layer1_Weights_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);
    double *Layer1_Mean_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double *Layer1_StanDev_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double *Layer1_Gamma_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double *Layer1_Beta_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);

    Read_First_Layer_Data(
        Layer1_Neurons_CPU,
        Layer1_Weights_CPU,        
        Layer1_Mean_CPU,
        Layer1_StanDev_CPU,
        Layer1_Gamma_CPU,
        Layer1_Beta_CPU
    );

    // Copy memory from Host to Kernel
    double *Layer1_Weights_GPU,
           *Layer1_Neurons_GPU,
           *Layer1_Mean_GPU,
           *Layer1_StanDev_GPU,
           *Layer1_Gamma_GPU,
           *Layer1_Beta_GPU;

    hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(double) * INPUT_LAYER_SIZE);
    hipMalloc((void**) &Layer1_Weights_GPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer1_Mean_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_StanDev_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_Gamma_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_Beta_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);

    hipMemcpy(Layer1_Neurons_GPU, Layer1_Neurons_CPU, sizeof(double) * INPUT_LAYER_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Weights_GPU, Layer1_Weights_CPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Mean_GPU, Layer1_Mean_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_StanDev_GPU, Layer1_StanDev_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Gamma_GPU, Layer1_Gamma_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Beta_GPU, Layer1_Beta_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer1_Neurons_CPU);
    free(Layer1_Weights_CPU);
    free(Layer1_Mean_CPU);
    free(Layer1_StanDev_CPU);
    free(Layer1_Gamma_CPU);
    free(Layer1_Beta_CPU);

    // Kernel Launch
    dim3 gridSizeA(32, 3, 3);
    dim3 blockSizeA(32,32);

    executeFirstLayer_partA<<< gridSizeA, blockSizeA>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );
    
    dim3 gridSizeB(32, 7);
    dim3 blockSizeB(16, 16);

    executeFirstLayer_partB<<< gridSizeB, blockSizeB>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );

    dim3 gridSizeC(32, 6);
    dim3 blockSizeC(16, 16);

    executeFirstLayer_partC<<< gridSizeC, blockSizeC>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );

    hipDeviceSynchronize();

    // First Layer GPU Memory Free
    hipFree(Layer1_Neurons_GPU);
    hipFree(Layer1_Weights_GPU);
    hipFree(Layer1_Mean_GPU);
    hipFree(Layer1_StanDev_GPU);
    hipFree(Layer1_Gamma_GPU);
    hipFree(Layer1_Beta_GPU);
}

void Read_First_Layer_Data(
    double * Layer1_Neurons_CPU,
    double * Layer1_Weights_CPU,
    double * Layer1_Mean_CPU,
    double * Layer1_StanDev_CPU,
    double * Layer1_Gamma_CPU,
    double * Layer1_Beta_CPU
){
    read_Input_File("data/FirstLayer/InputFiles/inputNorm.txt", Layer1_Neurons_CPU);
    read_File("data/FirstLayer/weightsNorm.txt", Layer1_Weights_CPU);
    read_File("data/FirstLayer/First_Layer_Mean.txt", Layer1_Mean_CPU);
    read_File("data/FirstLayer/First_Layer_StanDev.txt", Layer1_StanDev_CPU);
    read_File("data/FirstLayer/First_Layer_Gamma.txt", Layer1_Gamma_CPU);
    read_File("data/FirstLayer/First_Layer_Beta.txt", Layer1_Beta_CPU);
}

void Execute_Second_Layer(
    double * Layer2_Neurons_GPU,
    double * Layer3_Neurons_GPU
)
{
    double * Layer2_Weights_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_WEIGHT_SIZE);
    double * Layer2_Mean_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);
    double * Layer2_StanDev_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);
    double * Layer2_Gamma_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);
    double * Layer2_Beta_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);


    Read_SecondLayer_Data(Layer2_Weights_CPU,
                        Layer2_Mean_CPU,
                        Layer2_StanDev_CPU,
                        Layer2_Gamma_CPU,
                        Layer2_Beta_CPU
    );
    
    double *Layer2_Weights_GPU,
           *Layer2_Mean_GPU,
           *Layer2_StanDev_GPU,
           *Layer2_Gamma_GPU,
           *Layer2_Beta_GPU;;

    hipMalloc((void**) &Layer2_Weights_GPU, sizeof(double) * SECOND_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer2_Mean_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);
    hipMalloc((void**) &Layer2_StanDev_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);
    hipMalloc((void**) &Layer2_Gamma_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);
    hipMalloc((void**) &Layer2_Beta_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);

    hipMemcpy(Layer2_Weights_GPU, Layer2_Weights_CPU, sizeof(double) * SECOND_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Mean_GPU, Layer2_Mean_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_StanDev_GPU, Layer2_StanDev_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Gamma_GPU, Layer2_Gamma_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Beta_GPU, Layer2_Beta_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer2_Weights_CPU);
    free(Layer2_Mean_CPU);
    free(Layer2_StanDev_CPU);
    free(Layer2_Gamma_CPU);
    free(Layer2_Beta_CPU);

    dim3 gridSizeA(32, 3, 3);
    dim3 blockSizeA(32,32);
    executeSecondLayer_partA<<< gridSizeA, blockSizeA>>>(Layer2_Neurons_GPU,
                                            Layer2_Weights_GPU,
                                            Layer3_Neurons_GPU,
                                            Layer2_Mean_GPU,
                                            Layer2_StanDev_GPU,
                                            Layer2_Gamma_GPU,
                                            Layer2_Beta_GPU
    );

    dim3 gridSizeB(32, 7);
    dim3 blockSizeB(16, 16);
    executeSecondLayer_partB<<< gridSizeB, blockSizeB>>>(Layer2_Neurons_GPU,
                                            Layer2_Weights_GPU,
                                            Layer3_Neurons_GPU,
                                            Layer2_Mean_GPU,
                                            Layer2_StanDev_GPU,
                                            Layer2_Gamma_GPU,
                                            Layer2_Beta_GPU
    );

    dim3 gridSizeC(32, 6);
    dim3 blockSizeC(16, 16);
    executeSecondLayer_partC<<< gridSizeC, blockSizeC>>>(Layer2_Neurons_GPU,
                                            Layer2_Weights_GPU,
                                            Layer3_Neurons_GPU,
                                            Layer2_Mean_GPU,
                                            Layer2_StanDev_GPU,
                                            Layer2_Gamma_GPU,
                                            Layer2_Beta_GPU
    );

    hipFree(Layer2_Weights_GPU);    
    hipFree(Layer2_Mean_GPU);
    hipFree(Layer2_StanDev_GPU);
    hipFree(Layer2_Gamma_GPU);
    hipFree(Layer2_Beta_GPU);
}

void Read_SecondLayer_Data(double *Layer2_Weights_CPU,
    double * Layer2_Mean_CPU,
    double * Layer2_StanDev_CPU,
    double * Layer2_Gamma_CPU,
    double * Layer2_Beta_CPU
){
    read_File("data/SecondLayer/weightsNorm.txt", Layer2_Weights_CPU);
    read_File("data/SecondLayer/Second_Layer_Mean.txt", Layer2_Mean_CPU);
    read_File("data/SecondLayer/Second_Layer_StanDev.txt", Layer2_StanDev_CPU);
    read_File("data/SecondLayer/Second_Layer_Gamma.txt", Layer2_Gamma_CPU);
    read_File("data/SecondLayer/Second_Layer_Beta.txt", Layer2_Beta_CPU);
}

void Execute_Third_Layer(
    double * Layer3_Neurons_GPU,
    double * Layer4_Neurons_GPU
){
    double * Layer3_Weights_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_WEIGHT_SIZE);
    double * Layer3_Mean_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);
    double * Layer3_StanDev_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);
    double * Layer3_Gamma_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);
    double * Layer3_Beta_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);

    Read_ThirdLayer_Data(Layer3_Weights_CPU,
                Layer3_Mean_CPU,
                Layer3_StanDev_CPU,
                Layer3_Gamma_CPU,
                Layer3_Beta_CPU
    );

    double *Layer3_Weights_GPU,
           *Layer3_Mean_GPU,
           *Layer3_StanDev_GPU,
           *Layer3_Gamma_GPU,
           *Layer3_Beta_GPU;

    hipMalloc((void**) &Layer3_Weights_GPU, sizeof(double) * THIRD_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer3_Mean_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);
    hipMalloc((void**) &Layer3_StanDev_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);
    hipMalloc((void**) &Layer3_Gamma_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);
    hipMalloc((void**) &Layer3_Beta_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);

    hipMemcpy(Layer3_Weights_GPU, Layer3_Weights_CPU, sizeof(double) * THIRD_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Mean_GPU, Layer3_Mean_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_StanDev_GPU, Layer3_StanDev_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Gamma_GPU, Layer3_Gamma_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Beta_GPU, Layer3_Beta_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer3_Weights_CPU);
    free(Layer3_Mean_CPU);
    free(Layer3_StanDev_CPU);
    free(Layer3_Gamma_CPU);
    free(Layer3_Beta_CPU);
    
    // Execution of the Third Layer
    dim3 gridSizeThirdLayerA(64, 3, 3);
    dim3 blockSizeThirdLayerA(32,32);
    executeThirdLayer_partA<<< gridSizeThirdLayerA, blockSizeThirdLayerA>>>(Layer3_Neurons_GPU,
                        Layer3_Weights_GPU,
                        Layer4_Neurons_GPU,
                        Layer3_Mean_GPU,
                        Layer3_StanDev_GPU,
                        Layer3_Gamma_GPU,
                        Layer3_Beta_GPU
    );

    dim3 gridSizeThirdLayerB(64, 7);
    dim3 blockSizeThirdLayerB(16, 16);
    executeThirdLayer_partB<<< gridSizeThirdLayerB, blockSizeThirdLayerB>>>(Layer3_Neurons_GPU,
                        Layer3_Weights_GPU,
                        Layer4_Neurons_GPU,
                        Layer3_Mean_GPU,
                        Layer3_StanDev_GPU,
                        Layer3_Gamma_GPU,
                        Layer3_Beta_GPU
    );

    dim3 gridSizeThirdLayerC(64, 6);
    dim3 blockSizeThirdLayerC(16, 16);
    executeThirdLayer_partC<<< gridSizeThirdLayerC, blockSizeThirdLayerC>>>(Layer3_Neurons_GPU,
                        Layer3_Weights_GPU,
                        Layer4_Neurons_GPU,
                        Layer3_Mean_GPU,
                        Layer3_StanDev_GPU,
                        Layer3_Gamma_GPU,
                        Layer3_Beta_GPU
    );

    hipDeviceSynchronize();

    hipFree(Layer3_Weights_GPU);
    hipFree(Layer3_Mean_GPU);
    hipFree(Layer3_StanDev_GPU);
    hipFree(Layer3_Gamma_GPU);
    hipFree(Layer3_Beta_GPU);
}

void Read_ThirdLayer_Data(double *Layer3_Weights_CPU,
    double * Layer3_Mean_CPU,
    double * Layer3_StanDev_CPU,
    double * Layer3_Gamma_CPU,
    double * Layer3_Beta_CPU
){
    read_File("data/ThirdLayer/weightsNorm.txt", Layer3_Weights_CPU);
    read_File("data/ThirdLayer/Third_Layer_Mean.txt", Layer3_Mean_CPU);
    read_File("data/ThirdLayer/Third_Layer_StanDev.txt", Layer3_StanDev_CPU);
    read_File("data/ThirdLayer/Third_Layer_Gamma.txt", Layer3_Gamma_CPU);
    read_File("data/ThirdLayer/Third_Layer_Beta.txt", Layer3_Beta_CPU);
}

void Execute_Fourth_Layer(
    double * Layer4_Neurons_GPU,
    double * Layer5_Neurons_GPU
){  
    double * Layer4_Weights_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_WEIGHT_SIZE);
    double * Layer4_Mean_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);
    double * Layer4_StanDev_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);
    double * Layer4_Gamma_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);
    double * Layer4_Beta_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);

    Read_FourthLayer_Data(Layer4_Weights_CPU,
                    Layer4_Mean_CPU,
                    Layer4_StanDev_CPU,
                    Layer4_Gamma_CPU,
                    Layer4_Beta_CPU
    );
    
    double *Layer4_Weights_GPU,
           *Layer4_Mean_GPU,
           *Layer4_StanDev_GPU,
           *Layer4_Gamma_GPU,
           *Layer4_Beta_GPU;

    hipMalloc((void**) &Layer4_Weights_GPU, sizeof(double) * FOURTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer4_Mean_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer4_StanDev_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer4_Gamma_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer4_Beta_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);

    hipMemcpy(Layer4_Weights_GPU, Layer4_Weights_CPU, sizeof(double) * FOURTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Mean_GPU, Layer4_Mean_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_StanDev_GPU, Layer4_StanDev_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Gamma_GPU, Layer4_Gamma_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Beta_GPU, Layer4_Beta_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer4_Weights_CPU);
    free(Layer4_Mean_CPU);
    free(Layer4_StanDev_CPU);
    free(Layer4_Gamma_CPU);
    free(Layer4_Beta_CPU);

    dim3 gridSizeFourthLayer(64);
    dim3 blockSizeFourthLayerA(32,32);
    executeFourthLayer_partA<<< gridSizeFourthLayer, blockSizeFourthLayerA>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    dim3 blockSizeFourthLayerB(32, 24);
    executeFourthLayer_partB<<< gridSizeFourthLayer, blockSizeFourthLayerB>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    
    dim3 blockSizeFourthLayerC(24, 32);
    executeFourthLayer_partC<<< gridSizeFourthLayer, blockSizeFourthLayerC>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    
    dim3 blockSizeFourthLayerD(24, 24);
    executeFourthLayer_partD<<< gridSizeFourthLayer, blockSizeFourthLayerD>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    hipFree(Layer4_Weights_GPU);
    hipFree(Layer4_Mean_GPU);
    hipFree(Layer4_StanDev_GPU);
    hipFree(Layer4_Gamma_GPU);
    hipFree(Layer4_Beta_GPU);
}

void Read_FourthLayer_Data(double *Layer4_Weights_CPU,
    double * Layer4_Mean_CPU,
    double * Layer4_StanDev_CPU,
    double * Layer4_Gamma_CPU,
    double * Layer4_Beta_CPU
){
    read_File("data/FourthLayer/weightsNorm.txt", Layer4_Weights_CPU);
    read_File("data/FourthLayer/Fourth_Layer_Mean.txt", Layer4_Mean_CPU);
    read_File("data/FourthLayer/Fourth_Layer_StanDev.txt", Layer4_StanDev_CPU);
    read_File("data/FourthLayer/Fourth_Layer_Gamma.txt", Layer4_Gamma_CPU);
    read_File("data/FourthLayer/Fourth_Layer_Beta.txt", Layer4_Beta_CPU);
}

void Execute_Fifth_Layer(
    double * Layer5_Neurons_GPU,
    double * Layer6_Neurons_GPU
){  
    double * Layer5_Weights_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_WEIGHT_SIZE);
    double * Layer5_Mean_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_CHANNELS);
    double * Layer5_StanDev_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_CHANNELS);
    double * Layer5_Gamma_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_CHANNELS);
    double * Layer5_Beta_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_CHANNELS);

    Read_FifthLayer_Data(Layer5_Weights_CPU,
                    Layer5_Mean_CPU,
                    Layer5_StanDev_CPU,
                    Layer5_Gamma_CPU,
                    Layer5_Beta_CPU
                );
    
    double *Layer5_Weights_GPU,
           *Layer5_Mean_GPU,
           *Layer5_StanDev_GPU,
           *Layer5_Gamma_GPU,
           *Layer5_Beta_GPU;

    hipMalloc((void**) &Layer5_Weights_GPU, sizeof(double) * FIFTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer5_Mean_GPU, sizeof(double) * FIFTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer5_StanDev_GPU, sizeof(double) * FIFTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer5_Gamma_GPU, sizeof(double) * FIFTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer5_Beta_GPU, sizeof(double) * FIFTH_LAYER_CHANNELS);

    hipMemcpy(Layer5_Weights_GPU, Layer5_Weights_CPU, sizeof(double) * FIFTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer5_Mean_GPU, Layer5_Mean_CPU, sizeof(double) * FIFTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer5_StanDev_GPU, Layer5_StanDev_CPU, sizeof(double) * FIFTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer5_Gamma_GPU, Layer5_Gamma_CPU, sizeof(double) * FIFTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer5_Beta_GPU, Layer5_Beta_CPU, sizeof(double) * FIFTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer5_Weights_CPU);
    free(Layer5_Mean_CPU);
    free(Layer5_StanDev_CPU);
    free(Layer5_Gamma_CPU);
    free(Layer5_Beta_CPU);

    dim3 gridSizeFifthLayer(128);
    dim3 blockSizeFifthLayerA(32,32);
    executeFifthLayer_partA<<< gridSizeFifthLayer, blockSizeFifthLayerA>>>(Layer5_Neurons_GPU,
                        Layer5_Weights_GPU,
                        Layer6_Neurons_GPU,
                        Layer5_Mean_GPU,
                        Layer5_StanDev_GPU,
                        Layer5_Gamma_GPU,
                        Layer5_Beta_GPU
                    );
                    
    dim3 blockSizeFifthLayerB(32, 24);
    executeFifthLayer_partB<<< gridSizeFifthLayer, blockSizeFifthLayerB>>>(Layer5_Neurons_GPU,
                        Layer5_Weights_GPU,
                        Layer6_Neurons_GPU,
                        Layer5_Mean_GPU,
                        Layer5_StanDev_GPU,
                        Layer5_Gamma_GPU,
                        Layer5_Beta_GPU
                    );

    
    dim3 blockSizeFifthLayerC(24, 32);
    executeFifthLayer_partC<<< gridSizeFifthLayer, blockSizeFifthLayerC>>>(Layer5_Neurons_GPU,
                        Layer5_Weights_GPU,
                        Layer6_Neurons_GPU,
                        Layer5_Mean_GPU,
                        Layer5_StanDev_GPU,
                        Layer5_Gamma_GPU,
                        Layer5_Beta_GPU
                    );

    
    dim3 blockSizeFifthLayerD(24, 24);
    executeFifthLayer_partD<<< gridSizeFifthLayer, blockSizeFifthLayerD>>>(Layer5_Neurons_GPU,
                        Layer5_Weights_GPU,
                        Layer6_Neurons_GPU,
                        Layer5_Mean_GPU,
                        Layer5_StanDev_GPU,
                        Layer5_Gamma_GPU,
                        Layer5_Beta_GPU
                    );

    hipFree(Layer5_Weights_GPU);
    hipFree(Layer5_Mean_GPU);
    hipFree(Layer5_StanDev_GPU);
    hipFree(Layer5_Gamma_GPU);
    hipFree(Layer5_Beta_GPU);
}

void Read_FifthLayer_Data(double *Layer5_Weights_CPU,
    double * Layer5_Mean_CPU,
    double * Layer5_StanDev_CPU,
    double * Layer5_Gamma_CPU,
    double * Layer5_Beta_CPU

){
    read_File("data/FifthLayer/weightsNorm.txt", Layer5_Weights_CPU);
    read_File("data/FifthLayer/Fifth_Layer_Mean.txt", Layer5_Mean_CPU);
    read_File("data/FifthLayer/Fifth_Layer_StanDev.txt", Layer5_StanDev_CPU);
    read_File("data/FifthLayer/Fifth_Layer_Gamma.txt", Layer5_Gamma_CPU);
    read_File("data/FifthLayer/Fifth_Layer_Beta.txt", Layer5_Beta_CPU);
}

void Execute_Sixth_Layer(
    double * Layer6_Neurons_GPU,
    double * Layer7_Neurons_GPU
){  
    double * Layer6_Weights_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_WEIGHT_SIZE);
    double * Layer6_Mean_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_CHANNELS);
    double * Layer6_StanDev_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_CHANNELS);
    double * Layer6_Gamma_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_CHANNELS);
    double * Layer6_Beta_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_CHANNELS);

    Read_SixthLayer_Data(Layer6_Weights_CPU,
                    Layer6_Mean_CPU,
                    Layer6_StanDev_CPU,
                    Layer6_Gamma_CPU,
                    Layer6_Beta_CPU
                );
    
    double *Layer6_Weights_GPU,
           *Layer6_Mean_GPU,
           *Layer6_StanDev_GPU,
           *Layer6_Gamma_GPU,
           *Layer6_Beta_GPU;

    hipMalloc((void**) &Layer6_Weights_GPU, sizeof(double) * SIXTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer6_Mean_GPU, sizeof(double) * SIXTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer6_StanDev_GPU, sizeof(double) * SIXTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer6_Gamma_GPU, sizeof(double) * SIXTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer6_Beta_GPU, sizeof(double) * SIXTH_LAYER_CHANNELS);

    hipMemcpy(Layer6_Weights_GPU, Layer6_Weights_CPU, sizeof(double) * SIXTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer6_Mean_GPU, Layer6_Mean_CPU, sizeof(double) * SIXTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer6_StanDev_GPU, Layer6_StanDev_CPU, sizeof(double) * SIXTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer6_Gamma_GPU, Layer6_Gamma_CPU, sizeof(double) * SIXTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer6_Beta_GPU, Layer6_Beta_CPU, sizeof(double) * SIXTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer6_Weights_CPU);
    free(Layer6_Mean_CPU);
    free(Layer6_StanDev_CPU);
    free(Layer6_Gamma_CPU);
    free(Layer6_Beta_CPU);

    dim3 gridSizeSixthLayer(128);
    dim3 blockSizeSixthLayerA(32,32);
    executeSixthLayer_partA<<< gridSizeSixthLayer, blockSizeSixthLayerA>>>(Layer6_Neurons_GPU,
                        Layer6_Weights_GPU,
                        Layer7_Neurons_GPU,
                        Layer6_Mean_GPU,
                        Layer6_StanDev_GPU,
                        Layer6_Gamma_GPU,
                        Layer6_Beta_GPU
                    );
                    
    dim3 blockSizeSixthLayerB(32, 24);
    executeSixthLayer_partB<<< gridSizeSixthLayer, blockSizeSixthLayerB>>>(Layer6_Neurons_GPU,
                        Layer6_Weights_GPU,
                        Layer7_Neurons_GPU,
                        Layer6_Mean_GPU,
                        Layer6_StanDev_GPU,
                        Layer6_Gamma_GPU,
                        Layer6_Beta_GPU
                    );
    
    dim3 blockSizeSixthLayerC(24, 32);
    executeSixthLayer_partC<<< gridSizeSixthLayer, blockSizeSixthLayerC>>>(Layer6_Neurons_GPU,
                        Layer6_Weights_GPU,
                        Layer7_Neurons_GPU,
                        Layer6_Mean_GPU,
                        Layer6_StanDev_GPU,
                        Layer6_Gamma_GPU,
                        Layer6_Beta_GPU
                    );

    
    dim3 blockSizeSixthLayerD(24, 24);
    executeSixthLayer_partD<<< gridSizeSixthLayer, blockSizeSixthLayerD>>>(Layer6_Neurons_GPU,
                        Layer6_Weights_GPU,
                        Layer7_Neurons_GPU,
                        Layer6_Mean_GPU,
                        Layer6_StanDev_GPU,
                        Layer6_Gamma_GPU,
                        Layer6_Beta_GPU
                    );

    hipFree(Layer6_Weights_GPU);
    hipFree(Layer6_Mean_GPU);
    hipFree(Layer6_StanDev_GPU);
    hipFree(Layer6_Gamma_GPU);
    hipFree(Layer6_Beta_GPU);
}

void Read_SixthLayer_Data(double *Layer6_Weights_CPU,
    double * Layer6_Mean_CPU,
    double * Layer6_StanDev_CPU,
    double * Layer6_Gamma_CPU,
    double * Layer6_Beta_CPU
){
    read_File("data/SixthLayer/weightsNorm.txt", Layer6_Weights_CPU);
    read_File("data/SixthLayer/Sixth_Layer_Mean.txt", Layer6_Mean_CPU);
    read_File("data/SixthLayer/Sixth_Layer_StanDev.txt", Layer6_StanDev_CPU);
    read_File("data/SixthLayer/Sixth_Layer_Gamma.txt", Layer6_Gamma_CPU);
    read_File("data/SixthLayer/Sixth_Layer_Beta.txt", Layer6_Beta_CPU);
}

void Execute_Seventh_Layer(
    double * Layer7_Neurons_GPU,
    double * Layer8_Neurons_GPU
){  
    double * Layer7_Weights_CPU = (double *) malloc(sizeof(double) * SEVENTH_LAYER_WEIGHT_SIZE);
    double * Layer7_Mean_CPU = (double *) malloc(sizeof(double) * SEVENTH_LAYER_CHANNELS);
    double * Layer7_StanDev_CPU = (double *) malloc(sizeof(double) * SEVENTH_LAYER_CHANNELS);
    double * Layer7_Gamma_CPU = (double *) malloc(sizeof(double) * SEVENTH_LAYER_CHANNELS);
    double * Layer7_Beta_CPU = (double *) malloc(sizeof(double) * SEVENTH_LAYER_CHANNELS);

    Read_SeventhLayer_Data(Layer7_Weights_CPU,
                    Layer7_Mean_CPU,
                    Layer7_StanDev_CPU,
                    Layer7_Gamma_CPU,
                    Layer7_Beta_CPU
                );
    
    double *Layer7_Weights_GPU,
           *Layer7_Mean_GPU,
           *Layer7_StanDev_GPU,
           *Layer7_Gamma_GPU,
           *Layer7_Beta_GPU;

    hipMalloc((void**) &Layer7_Weights_GPU, sizeof(double) * SEVENTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer7_Mean_GPU, sizeof(double) * SEVENTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer7_StanDev_GPU, sizeof(double) * SEVENTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer7_Gamma_GPU, sizeof(double) * SEVENTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer7_Beta_GPU, sizeof(double) * SEVENTH_LAYER_CHANNELS);

    hipMemcpy(Layer7_Weights_GPU, Layer7_Weights_CPU, sizeof(double) * SEVENTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer7_Mean_GPU, Layer7_Mean_CPU, sizeof(double) * SEVENTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer7_StanDev_GPU, Layer7_StanDev_CPU, sizeof(double) * SEVENTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer7_Gamma_GPU, Layer7_Gamma_CPU, sizeof(double) * SEVENTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer7_Beta_GPU, Layer7_Beta_CPU, sizeof(double) * SEVENTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer7_Weights_CPU);
    free(Layer7_Mean_CPU);
    free(Layer7_StanDev_CPU);
    free(Layer7_Gamma_CPU);
    free(Layer7_Beta_CPU);

    dim3 gridSizeSeventhLayer(128);
    dim3 blockSizeSeventhLayerA(32,32);
    executeSeventhLayer_partA<<< gridSizeSeventhLayer, blockSizeSeventhLayerA>>>(Layer7_Neurons_GPU,
                        Layer7_Weights_GPU,
                        Layer8_Neurons_GPU,
                        Layer7_Mean_GPU,
                        Layer7_StanDev_GPU,
                        Layer7_Gamma_GPU,
                        Layer7_Beta_GPU
                    );
                    
    dim3 blockSizeSeventhLayerB(32, 24);
    executeSeventhLayer_partB<<< gridSizeSeventhLayer, blockSizeSeventhLayerB>>>(Layer7_Neurons_GPU,
                        Layer7_Weights_GPU,
                        Layer8_Neurons_GPU,
                        Layer7_Mean_GPU,
                        Layer7_StanDev_GPU,
                        Layer7_Gamma_GPU,
                        Layer7_Beta_GPU
                    );

    
    dim3 blockSizeSeventhLayerC(24, 32);
    executeSeventhLayer_partC<<< gridSizeSeventhLayer, blockSizeSeventhLayerC>>>(Layer7_Neurons_GPU,
                        Layer7_Weights_GPU,
                        Layer8_Neurons_GPU,
                        Layer7_Mean_GPU,
                        Layer7_StanDev_GPU,
                        Layer7_Gamma_GPU,
                        Layer7_Beta_GPU
                    );
    
    dim3 blockSizeSeventhLayerD(24, 24);
    executeSeventhLayer_partD<<< gridSizeSeventhLayer, blockSizeSeventhLayerD>>>(Layer7_Neurons_GPU,
                        Layer7_Weights_GPU,
                        Layer8_Neurons_GPU,
                        Layer7_Mean_GPU,
                        Layer7_StanDev_GPU,
                        Layer7_Gamma_GPU,
                        Layer7_Beta_GPU
                    );

    hipFree(Layer7_Weights_GPU);
    hipFree(Layer7_Mean_GPU);
    hipFree(Layer7_StanDev_GPU);
    hipFree(Layer7_Gamma_GPU);
    hipFree(Layer7_Beta_GPU);
}

void Read_SeventhLayer_Data(double *Layer7_Weights_CPU,
    double * Layer7_Mean_CPU,
    double * Layer7_StanDev_CPU,
    double * Layer7_Gamma_CPU,
    double * Layer7_Beta_CPU
){
    read_File("data/SeventhLayer/weightsNorm.txt", Layer7_Weights_CPU);
    read_File("data/SeventhLayer/Seventh_Layer_Mean.txt", Layer7_Mean_CPU);
    read_File("data/SeventhLayer/Seventh_Layer_StanDev.txt", Layer7_StanDev_CPU);
    read_File("data/SeventhLayer/Seventh_Layer_Gamma.txt", Layer7_Gamma_CPU);
    read_File("data/SeventhLayer/Seventh_Layer_Beta.txt", Layer7_Beta_CPU);
}

void Execute_Eighth_Layer(
    double * Layer8_Neurons_GPU,
    double * Layer9_Neurons_GPU
){  
    double * Layer8_Weights_CPU = (double *) malloc(sizeof(double) * EIGHTH_LAYER_WEIGHT_SIZE);
    double * Layer8_Mean_CPU = (double *) malloc(sizeof(double) * EIGHTH_LAYER_CHANNELS);
    double * Layer8_StanDev_CPU = (double *) malloc(sizeof(double) * EIGHTH_LAYER_CHANNELS);
    double * Layer8_Gamma_CPU = (double *) malloc(sizeof(double) * EIGHTH_LAYER_CHANNELS);
    double * Layer8_Beta_CPU = (double *) malloc(sizeof(double) * EIGHTH_LAYER_CHANNELS);

    Read_EighthLayer_Data(Layer8_Weights_CPU,
                    Layer8_Mean_CPU,
                    Layer8_StanDev_CPU,
                    Layer8_Gamma_CPU,
                    Layer8_Beta_CPU
                );
    
    double *Layer8_Weights_GPU,
           *Layer8_Mean_GPU,
           *Layer8_StanDev_GPU,
           *Layer8_Gamma_GPU,
           *Layer8_Beta_GPU;

    hipMalloc((void**) &Layer8_Weights_GPU, sizeof(double) * EIGHTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer8_Mean_GPU, sizeof(double) * EIGHTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer8_StanDev_GPU, sizeof(double) * EIGHTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer8_Gamma_GPU, sizeof(double) * EIGHTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer8_Beta_GPU, sizeof(double) * EIGHTH_LAYER_CHANNELS);

    hipMemcpy(Layer8_Weights_GPU, Layer8_Weights_CPU, sizeof(double) * EIGHTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer8_Mean_GPU, Layer8_Mean_CPU, sizeof(double) * EIGHTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer8_StanDev_GPU, Layer8_StanDev_CPU, sizeof(double) * EIGHTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer8_Gamma_GPU, Layer8_Gamma_CPU, sizeof(double) * EIGHTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer8_Beta_GPU, Layer8_Beta_CPU, sizeof(double) * EIGHTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer8_Weights_CPU);
    free(Layer8_Mean_CPU);
    free(Layer8_StanDev_CPU);
    free(Layer8_Gamma_CPU);
    free(Layer8_Beta_CPU);

    dim3 gridSizeEighthLayer(128);
    dim3 blockSizeEighth(28,28);
    executeEighthLayer<<< gridSizeEighthLayer, blockSizeEighth>>>(Layer8_Neurons_GPU,
                        Layer8_Weights_GPU,
                        Layer9_Neurons_GPU,
                        Layer8_Mean_GPU,
                        Layer8_StanDev_GPU,
                        Layer8_Gamma_GPU,
                        Layer8_Beta_GPU
                    );
                    
    hipFree(Layer8_Weights_GPU);
    hipFree(Layer8_Mean_GPU);
    hipFree(Layer8_StanDev_GPU);
    hipFree(Layer8_Gamma_GPU);
    hipFree(Layer8_Beta_GPU);
}

void Read_EighthLayer_Data(double *Layer8_Weights_CPU,
    double * Layer8_Mean_CPU,
    double * Layer8_StanDev_CPU,
    double * Layer8_Gamma_CPU,
    double * Layer8_Beta_CPU
){
    read_File("data/EighthLayer/weightsNorm.txt", Layer8_Weights_CPU);
    read_File("data/EighthLayer/Eighth_Layer_Mean.txt", Layer8_Mean_CPU);
    read_File("data/EighthLayer/Eighth_Layer_StanDev.txt", Layer8_StanDev_CPU);
    read_File("data/EighthLayer/Eighth_Layer_Gamma.txt", Layer8_Gamma_CPU);
    read_File("data/EighthLayer/Eighth_Layer_Beta.txt", Layer8_Beta_CPU);
}


void Execute_Ninth_Layer(
    double * Layer9_Neurons_GPU,
    double * Layer10_Neurons_GPU
){  
    double * Layer9_Weights_CPU = (double *) malloc(sizeof(double) * NINTH_LAYER_WEIGHT_SIZE);
    double * Layer9_Mean_CPU = (double *) malloc(sizeof(double) * NINTH_LAYER_CHANNELS);
    double * Layer9_StanDev_CPU = (double *) malloc(sizeof(double) * NINTH_LAYER_CHANNELS);
    double * Layer9_Gamma_CPU = (double *) malloc(sizeof(double) * NINTH_LAYER_CHANNELS);
    double * Layer9_Beta_CPU = (double *) malloc(sizeof(double) * NINTH_LAYER_CHANNELS);

    Read_NinthLayer_Data(Layer9_Weights_CPU,
                    Layer9_Mean_CPU,
                    Layer9_StanDev_CPU,
                    Layer9_Gamma_CPU,
                    Layer9_Beta_CPU
                );
    
    double *Layer9_Weights_GPU,
           *Layer9_Mean_GPU,
           *Layer9_StanDev_GPU,
           *Layer9_Gamma_GPU,
           *Layer9_Beta_GPU;

    hipMalloc((void**) &Layer9_Weights_GPU, sizeof(double) * NINTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer9_Mean_GPU, sizeof(double) * NINTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer9_StanDev_GPU, sizeof(double) * NINTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer9_Gamma_GPU, sizeof(double) * NINTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer9_Beta_GPU, sizeof(double) * NINTH_LAYER_CHANNELS);

    hipMemcpy(Layer9_Weights_GPU, Layer9_Weights_CPU, sizeof(double) * NINTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer9_Mean_GPU, Layer9_Mean_CPU, sizeof(double) * NINTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer9_StanDev_GPU, Layer9_StanDev_CPU, sizeof(double) * NINTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer9_Gamma_GPU, Layer9_Gamma_CPU, sizeof(double) * NINTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer9_Beta_GPU, Layer9_Beta_CPU, sizeof(double) * NINTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer9_Weights_CPU);
    free(Layer9_Mean_CPU);
    free(Layer9_StanDev_CPU);
    free(Layer9_Gamma_CPU);
    free(Layer9_Beta_CPU);

    dim3 gridSizeNinthLayer(256);
    dim3 blockSizeNinth(28,28);
    executeNinthLayer<<< gridSizeNinthLayer, blockSizeNinth>>>(Layer9_Neurons_GPU,
                        Layer9_Weights_GPU,
                        Layer10_Neurons_GPU,
                        Layer9_Mean_GPU,
                        Layer9_StanDev_GPU,
                        Layer9_Gamma_GPU,
                        Layer9_Beta_GPU
                    );
                    
    hipFree(Layer9_Weights_GPU);
    hipFree(Layer9_Mean_GPU);
    hipFree(Layer9_StanDev_GPU);
    hipFree(Layer9_Gamma_GPU);
    hipFree(Layer9_Beta_GPU);
}

void Read_NinthLayer_Data(double *Layer9_Weights_CPU,
    double * Layer9_Mean_CPU,
    double * Layer9_StanDev_CPU,
    double * Layer9_Gamma_CPU,
    double * Layer9_Beta_CPU
){
    read_File("data/NinthLayer/weightsNorm.txt", Layer9_Weights_CPU);
    read_File("data/NinthLayer/Ninth_Layer_Mean.txt", Layer9_Mean_CPU);
    read_File("data/NinthLayer/Ninth_Layer_StanDev.txt", Layer9_StanDev_CPU);
    read_File("data/NinthLayer/Ninth_Layer_Gamma.txt", Layer9_Gamma_CPU);
    read_File("data/NinthLayer/Ninth_Layer_Beta.txt", Layer9_Beta_CPU);
}

void Execute_Tenth_Layer(
    double * Layer10_Neurons_GPU,
    double * Layer11_Neurons_GPU
){  
    double * Layer10_Weights_CPU = (double *) malloc(sizeof(double) * TENTH_LAYER_WEIGHT_SIZE);
    double * Layer10_Mean_CPU = (double *) malloc(sizeof(double) * TENTH_LAYER_CHANNELS);
    double * Layer10_StanDev_CPU = (double *) malloc(sizeof(double) * TENTH_LAYER_CHANNELS);
    double * Layer10_Gamma_CPU = (double *) malloc(sizeof(double) * TENTH_LAYER_CHANNELS);
    double * Layer10_Beta_CPU = (double *) malloc(sizeof(double) * TENTH_LAYER_CHANNELS);

    Read_TenthLayer_Data(Layer10_Weights_CPU,
                    Layer10_Mean_CPU,
                    Layer10_StanDev_CPU,
                    Layer10_Gamma_CPU,
                    Layer10_Beta_CPU
                );
    
    double *Layer10_Weights_GPU,
           *Layer10_Mean_GPU,
           *Layer10_StanDev_GPU,
           *Layer10_Gamma_GPU,
           *Layer10_Beta_GPU;

    hipMalloc((void**) &Layer10_Weights_GPU, sizeof(double) * TENTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer10_Mean_GPU, sizeof(double) * TENTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer10_StanDev_GPU, sizeof(double) * TENTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer10_Gamma_GPU, sizeof(double) * TENTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer10_Beta_GPU, sizeof(double) * TENTH_LAYER_CHANNELS);

    hipMemcpy(Layer10_Weights_GPU, Layer10_Weights_CPU, sizeof(double) * TENTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer10_Mean_GPU, Layer10_Mean_CPU, sizeof(double) * TENTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer10_StanDev_GPU, Layer10_StanDev_CPU, sizeof(double) * TENTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer10_Gamma_GPU, Layer10_Gamma_CPU, sizeof(double) * TENTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer10_Beta_GPU, Layer10_Beta_CPU, sizeof(double) * TENTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer10_Weights_CPU);
    free(Layer10_Mean_CPU);
    free(Layer10_StanDev_CPU);
    free(Layer10_Gamma_CPU);
    free(Layer10_Beta_CPU);

    dim3 gridSizeTenthLayer(256);
    dim3 blockSizeTenth(28,28);
    executeTenthLayer<<< gridSizeTenthLayer, blockSizeTenth>>>(Layer10_Neurons_GPU,
                        Layer10_Weights_GPU,
                        Layer11_Neurons_GPU,
                        Layer10_Mean_GPU,
                        Layer10_StanDev_GPU,
                        Layer10_Gamma_GPU,
                        Layer10_Beta_GPU
                    );
                    
    hipFree(Layer10_Weights_GPU);
    hipFree(Layer10_Mean_GPU);
    hipFree(Layer10_StanDev_GPU);
    hipFree(Layer10_Gamma_GPU);
    hipFree(Layer10_Beta_GPU);
}

void Read_TenthLayer_Data(double *Layer10_Weights_CPU,
    double * Layer10_Mean_CPU,
    double * Layer10_StanDev_CPU,
    double * Layer10_Gamma_CPU,
    double * Layer10_Beta_CPU
){
    read_File("data/TenthLayer/weightsNorm.txt", Layer10_Weights_CPU);
    read_File("data/TenthLayer/Tenth_Layer_Mean.txt", Layer10_Mean_CPU);
    read_File("data/TenthLayer/Tenth_Layer_StanDev.txt", Layer10_StanDev_CPU);
    read_File("data/TenthLayer/Tenth_Layer_Gamma.txt", Layer10_Gamma_CPU);
    read_File("data/TenthLayer/Tenth_Layer_Beta.txt", Layer10_Beta_CPU);
}

void Execute_Eleventh_Layer(
    double * Layer11_Neurons_GPU,
    double * Layer12_Neurons_GPU
){  
    double * Layer11_Weights_CPU = (double *) malloc(sizeof(double) * ELEVENTH_LAYER_WEIGHT_SIZE);
    double * Layer11_Mean_CPU = (double *) malloc(sizeof(double) * ELEVENTH_LAYER_CHANNELS);
    double * Layer11_StanDev_CPU = (double *) malloc(sizeof(double) * ELEVENTH_LAYER_CHANNELS);
    double * Layer11_Gamma_CPU = (double *) malloc(sizeof(double) * ELEVENTH_LAYER_CHANNELS);
    double * Layer11_Beta_CPU = (double *) malloc(sizeof(double) * ELEVENTH_LAYER_CHANNELS);

    Read_EleventhLayer_Data(Layer11_Weights_CPU,
                    Layer11_Mean_CPU,
                    Layer11_StanDev_CPU,
                    Layer11_Gamma_CPU,
                    Layer11_Beta_CPU
                );
    
    double *Layer11_Weights_GPU,
           *Layer11_Mean_GPU,
           *Layer11_StanDev_GPU,
           *Layer11_Gamma_GPU,
           *Layer11_Beta_GPU;

    hipMalloc((void**) &Layer11_Weights_GPU, sizeof(double) * ELEVENTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer11_Mean_GPU, sizeof(double) * ELEVENTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer11_StanDev_GPU, sizeof(double) * ELEVENTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer11_Gamma_GPU, sizeof(double) * ELEVENTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer11_Beta_GPU, sizeof(double) * ELEVENTH_LAYER_CHANNELS);

    hipMemcpy(Layer11_Weights_GPU, Layer11_Weights_CPU, sizeof(double) * ELEVENTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer11_Mean_GPU, Layer11_Mean_CPU, sizeof(double) * ELEVENTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer11_StanDev_GPU, Layer11_StanDev_CPU, sizeof(double) * ELEVENTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer11_Gamma_GPU, Layer11_Gamma_CPU, sizeof(double) * ELEVENTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer11_Beta_GPU, Layer11_Beta_CPU, sizeof(double) * ELEVENTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer11_Weights_CPU);
    free(Layer11_Mean_CPU);
    free(Layer11_StanDev_CPU);
    free(Layer11_Gamma_CPU);
    free(Layer11_Beta_CPU);

    dim3 gridSizeEleventhLayer(256);
    dim3 blockSizeEleventh(28,28);
    executeEleventhLayer<<< gridSizeEleventhLayer, blockSizeEleventh>>>(Layer11_Neurons_GPU,
                        Layer11_Weights_GPU,
                        Layer12_Neurons_GPU,
                        Layer11_Mean_GPU,
                        Layer11_StanDev_GPU,
                        Layer11_Gamma_GPU,
                        Layer11_Beta_GPU
                    );
                    
    hipFree(Layer11_Weights_GPU);
    hipFree(Layer11_Mean_GPU);
    hipFree(Layer11_StanDev_GPU);
    hipFree(Layer11_Gamma_GPU);
    hipFree(Layer11_Beta_GPU);
}

void Read_EleventhLayer_Data(double *Layer11_Weights_CPU,
    double * Layer11_Mean_CPU,
    double * Layer11_StanDev_CPU,
    double * Layer11_Gamma_CPU,
    double * Layer11_Beta_CPU
){
    read_File("data/EleventhLayer/weightsNorm.txt", Layer11_Weights_CPU);
    read_File("data/EleventhLayer/Eleventh_Layer_Mean.txt", Layer11_Mean_CPU);
    read_File("data/EleventhLayer/Eleventh_Layer_StanDev.txt", Layer11_StanDev_CPU);
    read_File("data/EleventhLayer/Eleventh_Layer_Gamma.txt", Layer11_Gamma_CPU);
    read_File("data/EleventhLayer/Eleventh_Layer_Beta.txt", Layer11_Beta_CPU);
}

void Execute_Twelveth_Layer(
    double * Layer12_Neurons_GPU,
    double * Layer13_Neurons_GPU
){  
    double * Layer12_Weights_CPU = (double *) malloc(sizeof(double) * TWELFTH_LAYER_WEIGHT_SIZE);
    double * Layer12_Mean_CPU = (double *) malloc(sizeof(double) * TWELFTH_LAYER_CHANNELS);
    double * Layer12_StanDev_CPU = (double *) malloc(sizeof(double) * TWELFTH_LAYER_CHANNELS);
    double * Layer12_Gamma_CPU = (double *) malloc(sizeof(double) * TWELFTH_LAYER_CHANNELS);
    double * Layer12_Beta_CPU = (double *) malloc(sizeof(double) * TWELFTH_LAYER_CHANNELS);

    Read_TwelvethLayer_Data(Layer12_Weights_CPU,
                    Layer12_Mean_CPU,
                    Layer12_StanDev_CPU,
                    Layer12_Gamma_CPU,
                    Layer12_Beta_CPU
                );
    
    double *Layer12_Weights_GPU,
           *Layer12_Mean_GPU,
           *Layer12_StanDev_GPU,
           *Layer12_Gamma_GPU,
           *Layer12_Beta_GPU;

    hipMalloc((void**) &Layer12_Weights_GPU, sizeof(double) * TWELFTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer12_Mean_GPU, sizeof(double) * TWELFTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer12_StanDev_GPU, sizeof(double) * TWELFTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer12_Gamma_GPU, sizeof(double) * TWELFTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer12_Beta_GPU, sizeof(double) * TWELFTH_LAYER_CHANNELS);

    hipMemcpy(Layer12_Weights_GPU, Layer12_Weights_CPU, sizeof(double) * TWELFTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer12_Mean_GPU, Layer12_Mean_CPU, sizeof(double) * TWELFTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer12_StanDev_GPU, Layer12_StanDev_CPU, sizeof(double) * TWELFTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer12_Gamma_GPU, Layer12_Gamma_CPU, sizeof(double) * TWELFTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer12_Beta_GPU, Layer12_Beta_CPU, sizeof(double) * TWELFTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer12_Weights_CPU);
    free(Layer12_Mean_CPU);
    free(Layer12_StanDev_CPU);
    free(Layer12_Gamma_CPU);
    free(Layer12_Beta_CPU);

    dim3 gridSizeTwelvethLayer(256);
    dim3 blockSizeTwelveth(14,14);
    executeTwelfthLayer<<< gridSizeTwelvethLayer, blockSizeTwelveth>>>(Layer12_Neurons_GPU,
                        Layer12_Weights_GPU,
                        Layer13_Neurons_GPU,
                        Layer12_Mean_GPU,
                        Layer12_StanDev_GPU,
                        Layer12_Gamma_GPU,
                        Layer12_Beta_GPU
                    );
                    
    hipFree(Layer12_Weights_GPU);
    hipFree(Layer12_Mean_GPU);
    hipFree(Layer12_StanDev_GPU);
    hipFree(Layer12_Gamma_GPU);
    hipFree(Layer12_Beta_GPU);
}

void Read_TwelvethLayer_Data(double *Layer12_Weights_CPU,
    double * Layer12_Mean_CPU,
    double * Layer12_StanDev_CPU,
    double * Layer12_Gamma_CPU,
    double * Layer12_Beta_CPU
){
    read_File("data/TwelvethLayer/weightsNorm.txt", Layer12_Weights_CPU);
    read_File("data/TwelvethLayer/Twelveth_Layer_Mean.txt", Layer12_Mean_CPU);
    read_File("data/TwelvethLayer/Twelveth_Layer_StanDev.txt", Layer12_StanDev_CPU);
    read_File("data/TwelvethLayer/Twelveth_Layer_Gamma.txt", Layer12_Gamma_CPU);
    read_File("data/TwelvethLayer/Twelveth_Layer_Beta.txt", Layer12_Beta_CPU);
}

void read_File(const char * input_FileName, double * input_values){

    FILE *fp = fopen(input_FileName, "r");
    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;

    while ((read = getline(&line, &len, fp)) != -1)
        input_values[counter++] = atof(line);
    fclose(fp);
}

void read_Input_File(const char * inputFileName, double * Layer1_Neurons_CPU){
    FILE *fp = fopen(inputFileName, "r");

    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;
    int index = 0;
    int lastRow = 0;

    while ((read = getline(&line, &len, fp)) != -1) {
        Layer1_Neurons_CPU[counter++] = atof(line);
        index++;
        // handle padding
        if (index == 224){
            Layer1_Neurons_CPU[counter++] = 0;
            index = 0;
            lastRow++;
            if(lastRow == 224){
                lastRow = 0;
                int temp = 0;
                while (temp < 225) {
                    Layer1_Neurons_CPU[counter++] = 0;
                    temp++;
                }
            }
        }
    }
    fclose(fp);
}