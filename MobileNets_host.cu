#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>

#include "MobileNets_kernel.cu"

#define INPUT_LAYER_SIZE 225 * 225 * 3
#define FIRST_LAYER_WEIGHT_SIZE 32 * 3 * 3 * 3
#define FIRST_LAYER_OUTPUT_SIZE 114 * 114 * 32
#define FIRST_LAYER_CHANNELS 32

#define SECOND_LAYER_WEIGHT_SIZE 32 * 3 * 3
#define SECOND_LAYER_OUTPUT_SIZE 112 * 112 * 32
#define SECOND_LAYER_CHANNELS 32

#define THIRD_LAYER_WEIGHT_SIZE 64 * 32
#define THIRD_LAYER_OUTPUT_SIZE 113 * 113 * 64
#define THIRD_LAYER_CHANNELS 64

#define FOURTH_LAYER_WEIGHT_SIZE 3 * 3 * 64
#define FOURTH_LAYER_OUTPUT_SIZE 56 * 56 * 64
#define FOURTH_LAYER_CHANNELS 64

#define FIFTH_LAYER_WEIGHT_SIZE 64 * 128
#define FIFTH_LAYER_OUTPUT_SIZE 58 * 58 * 128
#define FIFTH_LAYER_CHANNELS 128

#define SIXTH_LAYER_WEIGHT_SIZE 3 * 3 * 128
#define SIXTH_LAYER_OUTPUT_SIZE 56 * 56 * 128
#define SIXTH_LAYER_CHANNELS 128

// Function declarations
void NeuralNetwork();
void read_File(const char * weightFileName, double *Layer1_Weights_CPU);
void read_Input_File(const char * inputFileName, double *Layer1_Neurons_CPU);

void Read_First_Layer_Data(double * Layer1_Neurons_CPU,
    double * Layer1_Weights_CPU,
    double * Layer1_Mean_CPU,
    double * Layer1_StanDev_CPU,
    double * Layer1_Gamma_CPU,
    double * Layer1_Beta_CPU
);

void Execute_First_Layer(double * Layer2_Neurons_GPU);

void Read_SecondLayer_Data(double *Layer1_Weights_CPU,
    double *Layer2_Mean_CPU,
    double *Layer2_StanDev_CPU,
    double *Layer2_Gamma_CPU,
    double *Layer2_Beta_CPU
);

void Execute_Second_Layer(
    double * Layer2_Neurons_GPU,
    double * Layer3_Neurons_GPU
);

void Read_ThirdLayer_Data(double *Layer3_Weights_CPU,
    double * Layer3_Mean_CPU,
    double * Layer3_StanDev_CPU,
    double * Layer3_Gamma_CPU,
    double * Layer3_Beta_CPU
);

void Execute_Third_Layer(
    double * Layer3_Neurons_GPU,
    double * Layer4_Neurons_GPU
);

void Read_FourthLayer_Data(double *Layer4_Weights_CPU,
    double * Layer4_Mean_CPU,
    double * Layer4_StanDev_CPU,
    double * Layer4_Gamma_CPU,
    double * Layer4_Beta_CPU
);

void Execute_Fourth_Layer(
    double * Layer4_Neurons_GPU,
    double * Layer5_Neurons_GPU
);

void Read_FifthLayer_Data(double *Layer5_Weights_CPU,
    double * Layer5_Mean_CPU,
    double * Layer5_StanDev_CPU,
    double * Layer5_Gamma_CPU,
    double * Layer5_Beta_CPU
);

void Execute_Fifth_Layer(
    double * Layer5_Neurons_GPU,
    double * Layer6_Neurons_GPU
);

void Read_SixthLayer_Data(double *Layer6_Weights_CPU,
    double * Layer6_Mean_CPU,
    double * Layer6_StanDev_CPU,
    double * Layer6_Gamma_CPU,
    double * Layer6_Beta_CPU
);

void Execute_Sixth_Layer(
    double * Layer6_Neurons_GPU,
    double * Layer7_Neurons_GPU
);

int main(){
    NeuralNetwork();
}

void NeuralNetwork(){
    FILE *fOutput;
    int value;

    /* ************************************************ FIRST LAYER ******************************************************** */
    double *Layer2_Neurons_GPU = NULL; 
    hipMalloc((void**) &Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);

    Execute_First_Layer(Layer2_Neurons_GPU);

    // Saving output of the first layer: Initially Not Saved
    bool SAVE_FIRST_LAYER_WEIGHTS = false;
    if(SAVE_FIRST_LAYER_WEIGHTS){
        
        double *Layer2_Neurons_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer2_Neurons_CPU, Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/FirstLayer/output.txt", "w");
        value = FIRST_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer2_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer2_Neurons_CPU);
    }
    
    printf("\n Layer 1 Execution complete !!!");
    /* ************************************************ FIRST LAYER COMPLETE *********************************************** */

    /* ************************************************ SECOND LAYER ******************************************************** */
    double *Layer3_Neurons_GPU;
    hipMalloc((void**) &Layer3_Neurons_GPU, sizeof(double) * SECOND_LAYER_OUTPUT_SIZE);

    Execute_Second_Layer(Layer2_Neurons_GPU, Layer3_Neurons_GPU);

    bool SAVE_SECOND_LAYER_WEIGHTS = false;
    if(SAVE_SECOND_LAYER_WEIGHTS){
        
        double * Layer3_Neurons_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer3_Neurons_CPU, Layer3_Neurons_GPU, sizeof(double) * SECOND_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/SecondLayer/output.txt", "w");
        value = SECOND_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer3_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer3_Neurons_CPU);
    }

    printf("\n Layer 2 Execution complete !!!");
    /* ************************************************ SECOND LAYER COMPLETE *********************************************** */

    /* ************************************************ THIRD LAYER ******************************************************** */
    double *Layer4_Neurons_GPU;
    hipMalloc((void**) &Layer4_Neurons_GPU, sizeof(double) * THIRD_LAYER_OUTPUT_SIZE);

    Execute_Third_Layer(Layer3_Neurons_GPU, Layer4_Neurons_GPU);

    bool SAVE_THIRD_LAYER_WEIGHTS = true;
    if(SAVE_THIRD_LAYER_WEIGHTS){
        double * Layer4_Neurons_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer4_Neurons_CPU, Layer4_Neurons_GPU, sizeof(double) * THIRD_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/ThirdLayer/output.txt", "w");
        value = THIRD_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer4_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer4_Neurons_CPU);
    }

    printf("\n Layer 3 Execution complete !!!");
    /* ************************************************ THIRD LAYER COMPLETE *********************************************** */

    /* ************************************************ FOURTH LAYER ******************************************************** */
    double *Layer5_Neurons_GPU;
    hipMalloc((void**) &Layer5_Neurons_GPU, sizeof(double) * FOURTH_LAYER_OUTPUT_SIZE);

    Execute_Fourth_Layer(Layer4_Neurons_GPU, Layer5_Neurons_GPU);

    bool SAVE_FOURTH_LAYER_WEIGHTS = true;
    if(SAVE_FOURTH_LAYER_WEIGHTS){
        double * Layer5_Neurons_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer5_Neurons_CPU, Layer5_Neurons_GPU, sizeof(double) * FOURTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/FourthLayer/output.txt", "w");
        value = FOURTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer5_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer5_Neurons_CPU);
    }

    printf("\n Layer 4 Execution complete !!!");
    /* ************************************************ FOURTH LAYER COMPLETE *********************************************** */

    /* ************************************************ FIFTH LAYER ******************************************************** */
    double *Layer6_Neurons_GPU;
    hipMalloc((void**) &Layer6_Neurons_GPU, sizeof(double) * FIFTH_LAYER_OUTPUT_SIZE);

    Execute_Fifth_Layer(Layer5_Neurons_GPU, Layer6_Neurons_GPU);

    bool SAVE_FIFTH_LAYER_WEIGHTS = true;
    if(SAVE_FIFTH_LAYER_WEIGHTS){
        double * Layer6_Neurons_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer6_Neurons_CPU, Layer6_Neurons_GPU, sizeof(double) * FIFTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/FifthLayer/output.txt", "w");
        value = FIFTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer6_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer6_Neurons_CPU);
    }

    printf("\n Layer 5 Execution complete !!!");
    /* ************************************************ FIFTH LAYER COMPLETE *********************************************** */

    /* ************************************************ SIXTH LAYER ******************************************************** */
    double *Layer7_Neurons_GPU;
    hipMalloc((void**) &Layer7_Neurons_GPU, sizeof(double) * SIXTH_LAYER_OUTPUT_SIZE);

    Execute_Sixth_Layer(Layer6_Neurons_GPU, Layer7_Neurons_GPU);

    bool SAVE_SIXTH_LAYER_WEIGHTS = true;
    if(SAVE_SIXTH_LAYER_WEIGHTS){
        double * Layer7_Neurons_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer7_Neurons_CPU, Layer7_Neurons_GPU, sizeof(double) * SIXTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/SixthLayer/output.txt", "w");
        value = SIXTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer7_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer7_Neurons_CPU);
    }

    printf("\n Layer 6 Execution complete !!!");
    /* ************************************************ FIFTH LAYER COMPLETE *********************************************** */

    printf("\n\n Processing Done !!! \n\n");

    hipFree(Layer2_Neurons_GPU);
    hipFree(Layer3_Neurons_GPU);
    hipFree(Layer4_Neurons_GPU);
}

void Execute_First_Layer(double *Layer2_Neurons_GPU)
{
    double *Layer1_Neurons_CPU = (double *) malloc(sizeof(double) * INPUT_LAYER_SIZE);
    double *Layer1_Weights_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);
    double *Layer1_Mean_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double *Layer1_StanDev_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double *Layer1_Gamma_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double *Layer1_Beta_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);

    Read_First_Layer_Data(
        Layer1_Neurons_CPU,
        Layer1_Weights_CPU,        
        Layer1_Mean_CPU,
        Layer1_StanDev_CPU,
        Layer1_Gamma_CPU,
        Layer1_Beta_CPU
    );

    // Copy memory from Host to Kernel
    double *Layer1_Weights_GPU,
           *Layer1_Neurons_GPU,
           *Layer1_Mean_GPU,
           *Layer1_StanDev_GPU,
           *Layer1_Gamma_GPU,
           *Layer1_Beta_GPU;

    hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(double) * INPUT_LAYER_SIZE);
    hipMalloc((void**) &Layer1_Weights_GPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer1_Mean_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_StanDev_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_Gamma_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_Beta_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);

    hipMemcpy(Layer1_Neurons_GPU, Layer1_Neurons_CPU, sizeof(double) * INPUT_LAYER_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Weights_GPU, Layer1_Weights_CPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Mean_GPU, Layer1_Mean_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_StanDev_GPU, Layer1_StanDev_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Gamma_GPU, Layer1_Gamma_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Beta_GPU, Layer1_Beta_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer1_Neurons_CPU);
    free(Layer1_Weights_CPU);
    free(Layer1_Mean_CPU);
    free(Layer1_StanDev_CPU);
    free(Layer1_Gamma_CPU);
    free(Layer1_Beta_CPU);

    // Kernel Launch
    dim3 gridSizeA(32, 3, 3);
    dim3 blockSizeA(32,32);

    executeFirstLayer_partA<<< gridSizeA, blockSizeA>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );
    
    dim3 gridSizeB(32, 7);
    dim3 blockSizeB(16, 16);

    executeFirstLayer_partB<<< gridSizeB, blockSizeB>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );

    dim3 gridSizeC(32, 6);
    dim3 blockSizeC(16, 16);

    executeFirstLayer_partC<<< gridSizeC, blockSizeC>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );

    hipDeviceSynchronize();

    // First Layer GPU Memory Free
    hipFree(Layer1_Neurons_GPU);
    hipFree(Layer1_Weights_GPU);
    hipFree(Layer1_Mean_GPU);
    hipFree(Layer1_StanDev_GPU);
    hipFree(Layer1_Gamma_GPU);
    hipFree(Layer1_Beta_GPU);
}

void Read_First_Layer_Data(
    double * Layer1_Neurons_CPU,
    double * Layer1_Weights_CPU,
    double * Layer1_Mean_CPU,
    double * Layer1_StanDev_CPU,
    double * Layer1_Gamma_CPU,
    double * Layer1_Beta_CPU
){
    read_Input_File("data/FirstLayer/InputFiles/inputNorm.txt", Layer1_Neurons_CPU);
    read_File("data/FirstLayer/weightsNorm.txt", Layer1_Weights_CPU);
    read_File("data/FirstLayer/First_Layer_Mean.txt", Layer1_Mean_CPU);
    read_File("data/FirstLayer/First_Layer_StanDev.txt", Layer1_StanDev_CPU);
    read_File("data/FirstLayer/First_Layer_Gamma.txt", Layer1_Gamma_CPU);
    read_File("data/FirstLayer/First_Layer_Beta.txt", Layer1_Beta_CPU);
}

void Execute_Second_Layer(
    double * Layer2_Neurons_GPU,
    double * Layer3_Neurons_GPU
)
{
    double * Layer2_Weights_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_WEIGHT_SIZE);
    double * Layer2_Mean_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);
    double * Layer2_StanDev_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);
    double * Layer2_Gamma_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);
    double * Layer2_Beta_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);


    Read_SecondLayer_Data(Layer2_Weights_CPU,
                        Layer2_Mean_CPU,
                        Layer2_StanDev_CPU,
                        Layer2_Gamma_CPU,
                        Layer2_Beta_CPU
    );
    
    double *Layer2_Weights_GPU,
           *Layer2_Mean_GPU,
           *Layer2_StanDev_GPU,
           *Layer2_Gamma_GPU,
           *Layer2_Beta_GPU;;

    hipMalloc((void**) &Layer2_Weights_GPU, sizeof(double) * SECOND_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer2_Mean_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);
    hipMalloc((void**) &Layer2_StanDev_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);
    hipMalloc((void**) &Layer2_Gamma_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);
    hipMalloc((void**) &Layer2_Beta_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);

    hipMemcpy(Layer2_Weights_GPU, Layer2_Weights_CPU, sizeof(double) * SECOND_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Mean_GPU, Layer2_Mean_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_StanDev_GPU, Layer2_StanDev_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Gamma_GPU, Layer2_Gamma_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Beta_GPU, Layer2_Beta_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer2_Weights_CPU);
    free(Layer2_Mean_CPU);
    free(Layer2_StanDev_CPU);
    free(Layer2_Gamma_CPU);
    free(Layer2_Beta_CPU);

    dim3 gridSizeA(32, 3, 3);
    dim3 blockSizeA(32,32);
    executeSecondLayer_partA<<< gridSizeA, blockSizeA>>>(Layer2_Neurons_GPU,
                                            Layer2_Weights_GPU,
                                            Layer3_Neurons_GPU,
                                            Layer2_Mean_GPU,
                                            Layer2_StanDev_GPU,
                                            Layer2_Gamma_GPU,
                                            Layer2_Beta_GPU
    );

    dim3 gridSizeB(32, 7);
    dim3 blockSizeB(16, 16);
    executeSecondLayer_partB<<< gridSizeB, blockSizeB>>>(Layer2_Neurons_GPU,
                                            Layer2_Weights_GPU,
                                            Layer3_Neurons_GPU,
                                            Layer2_Mean_GPU,
                                            Layer2_StanDev_GPU,
                                            Layer2_Gamma_GPU,
                                            Layer2_Beta_GPU
    );

    dim3 gridSizeC(32, 6);
    dim3 blockSizeC(16, 16);
    executeSecondLayer_partC<<< gridSizeC, blockSizeC>>>(Layer2_Neurons_GPU,
                                            Layer2_Weights_GPU,
                                            Layer3_Neurons_GPU,
                                            Layer2_Mean_GPU,
                                            Layer2_StanDev_GPU,
                                            Layer2_Gamma_GPU,
                                            Layer2_Beta_GPU
    );

    hipFree(Layer2_Weights_GPU);    
    hipFree(Layer2_Mean_GPU);
    hipFree(Layer2_StanDev_GPU);
    hipFree(Layer2_Gamma_GPU);
    hipFree(Layer2_Beta_GPU);
}

void Read_SecondLayer_Data(double *Layer2_Weights_CPU,
    double * Layer2_Mean_CPU,
    double * Layer2_StanDev_CPU,
    double * Layer2_Gamma_CPU,
    double * Layer2_Beta_CPU
){
    read_File("data/SecondLayer/weightsNorm.txt", Layer2_Weights_CPU);
    read_File("data/SecondLayer/Second_Layer_Mean.txt", Layer2_Mean_CPU);
    read_File("data/SecondLayer/Second_Layer_StanDev.txt", Layer2_StanDev_CPU);
    read_File("data/SecondLayer/Second_Layer_Gamma.txt", Layer2_Gamma_CPU);
    read_File("data/SecondLayer/Second_Layer_Beta.txt", Layer2_Beta_CPU);
}

void Execute_Third_Layer(
    double * Layer3_Neurons_GPU,
    double * Layer4_Neurons_GPU
){
    double * Layer3_Weights_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_WEIGHT_SIZE);
    double * Layer3_Mean_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);
    double * Layer3_StanDev_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);
    double * Layer3_Gamma_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);
    double * Layer3_Beta_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);

    Read_ThirdLayer_Data(Layer3_Weights_CPU,
                Layer3_Mean_CPU,
                Layer3_StanDev_CPU,
                Layer3_Gamma_CPU,
                Layer3_Beta_CPU
    );

    double *Layer3_Weights_GPU,
           *Layer3_Mean_GPU,
           *Layer3_StanDev_GPU,
           *Layer3_Gamma_GPU,
           *Layer3_Beta_GPU;

    hipMalloc((void**) &Layer3_Weights_GPU, sizeof(double) * THIRD_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer3_Mean_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);
    hipMalloc((void**) &Layer3_StanDev_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);
    hipMalloc((void**) &Layer3_Gamma_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);
    hipMalloc((void**) &Layer3_Beta_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);

    hipMemcpy(Layer3_Weights_GPU, Layer3_Weights_CPU, sizeof(double) * THIRD_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Mean_GPU, Layer3_Mean_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_StanDev_GPU, Layer3_StanDev_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Gamma_GPU, Layer3_Gamma_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Beta_GPU, Layer3_Beta_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer3_Weights_CPU);
    free(Layer3_Mean_CPU);
    free(Layer3_StanDev_CPU);
    free(Layer3_Gamma_CPU);
    free(Layer3_Beta_CPU);
    
    // Execution of the Third Layer
    dim3 gridSizeThirdLayerA(64, 3, 3);
    dim3 blockSizeThirdLayerA(32,32);
    executeThirdLayer_partA<<< gridSizeThirdLayerA, blockSizeThirdLayerA>>>(Layer3_Neurons_GPU,
                        Layer3_Weights_GPU,
                        Layer4_Neurons_GPU,
                        Layer3_Mean_GPU,
                        Layer3_StanDev_GPU,
                        Layer3_Gamma_GPU,
                        Layer3_Beta_GPU
    );

    dim3 gridSizeThirdLayerB(64, 7);
    dim3 blockSizeThirdLayerB(16, 16);
    executeThirdLayer_partB<<< gridSizeThirdLayerB, blockSizeThirdLayerB>>>(Layer3_Neurons_GPU,
                        Layer3_Weights_GPU,
                        Layer4_Neurons_GPU,
                        Layer3_Mean_GPU,
                        Layer3_StanDev_GPU,
                        Layer3_Gamma_GPU,
                        Layer3_Beta_GPU
    );

    dim3 gridSizeThirdLayerC(64, 6);
    dim3 blockSizeThirdLayerC(16, 16);
    executeThirdLayer_partC<<< gridSizeThirdLayerC, blockSizeThirdLayerC>>>(Layer3_Neurons_GPU,
                        Layer3_Weights_GPU,
                        Layer4_Neurons_GPU,
                        Layer3_Mean_GPU,
                        Layer3_StanDev_GPU,
                        Layer3_Gamma_GPU,
                        Layer3_Beta_GPU
    );

    hipDeviceSynchronize();

    hipFree(Layer3_Weights_GPU);
    hipFree(Layer3_Mean_GPU);
    hipFree(Layer3_StanDev_GPU);
    hipFree(Layer3_Gamma_GPU);
    hipFree(Layer3_Beta_GPU);
}

void Read_ThirdLayer_Data(double *Layer3_Weights_CPU,
    double * Layer3_Mean_CPU,
    double * Layer3_StanDev_CPU,
    double * Layer3_Gamma_CPU,
    double * Layer3_Beta_CPU
){
    read_File("data/ThirdLayer/weightsNorm.txt", Layer3_Weights_CPU);
    read_File("data/ThirdLayer/Third_Layer_Mean.txt", Layer3_Mean_CPU);
    read_File("data/ThirdLayer/Third_Layer_StanDev.txt", Layer3_StanDev_CPU);
    read_File("data/ThirdLayer/Third_Layer_Gamma.txt", Layer3_Gamma_CPU);
    read_File("data/ThirdLayer/Third_Layer_Beta.txt", Layer3_Beta_CPU);
}

void Execute_Fourth_Layer(
    double * Layer4_Neurons_GPU,
    double * Layer5_Neurons_GPU
){  
    double * Layer4_Weights_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_WEIGHT_SIZE);
    double * Layer4_Mean_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);
    double * Layer4_StanDev_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);
    double * Layer4_Gamma_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);
    double * Layer4_Beta_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);

    Read_FourthLayer_Data(Layer4_Weights_CPU,
                    Layer4_Mean_CPU,
                    Layer4_StanDev_CPU,
                    Layer4_Gamma_CPU,
                    Layer4_Beta_CPU
    );
    
    double *Layer4_Weights_GPU,
           *Layer4_Mean_GPU,
           *Layer4_StanDev_GPU,
           *Layer4_Gamma_GPU,
           *Layer4_Beta_GPU;

    hipMalloc((void**) &Layer4_Weights_GPU, sizeof(double) * FOURTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer4_Mean_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer4_StanDev_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer4_Gamma_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer4_Beta_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);

    hipMemcpy(Layer4_Weights_GPU, Layer4_Weights_CPU, sizeof(double) * FOURTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Mean_GPU, Layer4_Mean_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_StanDev_GPU, Layer4_StanDev_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Gamma_GPU, Layer4_Gamma_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Beta_GPU, Layer4_Beta_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer4_Weights_CPU);
    free(Layer4_Mean_CPU);
    free(Layer4_StanDev_CPU);
    free(Layer4_Gamma_CPU);
    free(Layer4_Beta_CPU);

    dim3 gridSizeFourthLayer(64);
    dim3 blockSizeFourthLayerA(32,32);
    executeFourthLayer_partA<<< gridSizeFourthLayer, blockSizeFourthLayerA>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    dim3 blockSizeFourthLayerB(32, 24);
    executeFourthLayer_partB<<< gridSizeFourthLayer, blockSizeFourthLayerB>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    
    dim3 blockSizeFourthLayerC(24, 32);
    executeFourthLayer_partC<<< gridSizeFourthLayer, blockSizeFourthLayerC>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    
    dim3 blockSizeFourthLayerD(24, 24);
    executeFourthLayer_partD<<< gridSizeFourthLayer, blockSizeFourthLayerD>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    hipFree(Layer4_Weights_GPU);
    hipFree(Layer4_Mean_GPU);
    hipFree(Layer4_StanDev_GPU);
    hipFree(Layer4_Gamma_GPU);
    hipFree(Layer4_Beta_GPU);
}

void Read_FourthLayer_Data(double *Layer4_Weights_CPU,
    double * Layer4_Mean_CPU,
    double * Layer4_StanDev_CPU,
    double * Layer4_Gamma_CPU,
    double * Layer4_Beta_CPU
){
    read_File("data/FourthLayer/weightsNorm.txt", Layer4_Weights_CPU);
    read_File("data/FourthLayer/Fourth_Layer_Mean.txt", Layer4_Mean_CPU);
    read_File("data/FourthLayer/Fourth_Layer_StanDev.txt", Layer4_StanDev_CPU);
    read_File("data/FourthLayer/Fourth_Layer_Gamma.txt", Layer4_Gamma_CPU);
    read_File("data/FourthLayer/Fourth_Layer_Beta.txt", Layer4_Beta_CPU);
}

void Execute_Fifth_Layer(
    double * Layer5_Neurons_GPU,
    double * Layer6_Neurons_GPU
){  
    double * Layer5_Weights_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_WEIGHT_SIZE);
    double * Layer5_Mean_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_CHANNELS);
    double * Layer5_StanDev_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_CHANNELS);
    double * Layer5_Gamma_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_CHANNELS);
    double * Layer5_Beta_CPU = (double *) malloc(sizeof(double) * FIFTH_LAYER_CHANNELS);

    Read_FifthLayer_Data(Layer5_Weights_CPU,
                    Layer5_Mean_CPU,
                    Layer5_StanDev_CPU,
                    Layer5_Gamma_CPU,
                    Layer5_Beta_CPU
                );
    
    double *Layer5_Weights_GPU,
           *Layer5_Mean_GPU,
           *Layer5_StanDev_GPU,
           *Layer5_Gamma_GPU,
           *Layer5_Beta_GPU;

    hipMalloc((void**) &Layer5_Weights_GPU, sizeof(double) * FIFTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer5_Mean_GPU, sizeof(double) * FIFTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer5_StanDev_GPU, sizeof(double) * FIFTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer5_Gamma_GPU, sizeof(double) * FIFTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer5_Beta_GPU, sizeof(double) * FIFTH_LAYER_CHANNELS);

    hipMemcpy(Layer5_Weights_GPU, Layer5_Weights_CPU, sizeof(double) * FIFTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer5_Mean_GPU, Layer5_Mean_CPU, sizeof(double) * FIFTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer5_StanDev_GPU, Layer5_StanDev_CPU, sizeof(double) * FIFTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer5_Gamma_GPU, Layer5_Gamma_CPU, sizeof(double) * FIFTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer5_Beta_GPU, Layer5_Beta_CPU, sizeof(double) * FIFTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer5_Weights_CPU);
    free(Layer5_Mean_CPU);
    free(Layer5_StanDev_CPU);
    free(Layer5_Gamma_CPU);
    free(Layer5_Beta_CPU);

    dim3 gridSizeFifthLayer(128);
    dim3 blockSizeFifthLayerA(32,32);
    executeFifthLayer_partA<<< gridSizeFifthLayer, blockSizeFifthLayerA>>>(Layer5_Neurons_GPU,
                        Layer5_Weights_GPU,
                        Layer6_Neurons_GPU,
                        Layer5_Mean_GPU,
                        Layer5_StanDev_GPU,
                        Layer5_Gamma_GPU,
                        Layer5_Beta_GPU
                    );
                    
    dim3 blockSizeFifthLayerB(32, 24);
    executeFifthLayer_partB<<< gridSizeFifthLayer, blockSizeFifthLayerB>>>(Layer5_Neurons_GPU,
                        Layer5_Weights_GPU,
                        Layer6_Neurons_GPU,
                        Layer5_Mean_GPU,
                        Layer5_StanDev_GPU,
                        Layer5_Gamma_GPU,
                        Layer5_Beta_GPU
                    );

    
    dim3 blockSizeFifthLayerC(24, 32);
    executeFifthLayer_partC<<< gridSizeFifthLayer, blockSizeFifthLayerC>>>(Layer5_Neurons_GPU,
                        Layer5_Weights_GPU,
                        Layer6_Neurons_GPU,
                        Layer5_Mean_GPU,
                        Layer5_StanDev_GPU,
                        Layer5_Gamma_GPU,
                        Layer5_Beta_GPU
                    );

    
    dim3 blockSizeFifthLayerD(24, 24);
    executeFifthLayer_partD<<< gridSizeFifthLayer, blockSizeFifthLayerD>>>(Layer5_Neurons_GPU,
                        Layer5_Weights_GPU,
                        Layer6_Neurons_GPU,
                        Layer5_Mean_GPU,
                        Layer5_StanDev_GPU,
                        Layer5_Gamma_GPU,
                        Layer5_Beta_GPU
                    );

    hipFree(Layer5_Weights_GPU);
    hipFree(Layer5_Mean_GPU);
    hipFree(Layer5_StanDev_GPU);
    hipFree(Layer5_Gamma_GPU);
    hipFree(Layer5_Beta_GPU);
}

void Read_FifthLayer_Data(double *Layer5_Weights_CPU,
    double * Layer5_Mean_CPU,
    double * Layer5_StanDev_CPU,
    double * Layer5_Gamma_CPU,
    double * Layer5_Beta_CPU

){
    read_File("data/FifthLayer/weightsNorm.txt", Layer5_Weights_CPU);
    read_File("data/FifthLayer/Fifth_Layer_Mean.txt", Layer5_Mean_CPU);
    read_File("data/FifthLayer/Fifth_Layer_StanDev.txt", Layer5_StanDev_CPU);
    read_File("data/FifthLayer/Fifth_Layer_Gamma.txt", Layer5_Gamma_CPU);
    read_File("data/FifthLayer/Fifth_Layer_Beta.txt", Layer5_Beta_CPU);
}

void Execute_Sixth_Layer(
    double * Layer6_Neurons_GPU,
    double * Layer7_Neurons_GPU
){  
    double * Layer6_Weights_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_WEIGHT_SIZE);
    double * Layer6_Mean_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_CHANNELS);
    double * Layer6_StanDev_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_CHANNELS);
    double * Layer6_Gamma_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_CHANNELS);
    double * Layer6_Beta_CPU = (double *) malloc(sizeof(double) * SIXTH_LAYER_CHANNELS);

    Read_SixthLayer_Data(Layer6_Weights_CPU,
                    Layer6_Mean_CPU,
                    Layer6_StanDev_CPU,
                    Layer6_Gamma_CPU,
                    Layer6_Beta_CPU
                );
    
    double *Layer6_Weights_GPU,
           *Layer6_Mean_GPU,
           *Layer6_StanDev_GPU,
           *Layer6_Gamma_GPU,
           *Layer6_Beta_GPU;

    hipMalloc((void**) &Layer6_Weights_GPU, sizeof(double) * SIXTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer6_Mean_GPU, sizeof(double) * SIXTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer6_StanDev_GPU, sizeof(double) * SIXTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer6_Gamma_GPU, sizeof(double) * SIXTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer6_Beta_GPU, sizeof(double) * SIXTH_LAYER_CHANNELS);

    hipMemcpy(Layer6_Weights_GPU, Layer6_Weights_CPU, sizeof(double) * SIXTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer6_Mean_GPU, Layer6_Mean_CPU, sizeof(double) * SIXTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer6_StanDev_GPU, Layer6_StanDev_CPU, sizeof(double) * SIXTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer6_Gamma_GPU, Layer6_Gamma_CPU, sizeof(double) * SIXTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer6_Beta_GPU, Layer6_Beta_CPU, sizeof(double) * SIXTH_LAYER_CHANNELS, hipMemcpyHostToDevice); 

    free(Layer6_Weights_CPU);
    free(Layer6_Mean_CPU);
    free(Layer6_StanDev_CPU);
    free(Layer6_Gamma_CPU);
    free(Layer6_Beta_CPU);

    dim3 gridSizeSixthLayer(128);
    dim3 blockSizeSixthLayerA(32,32);
    executeSixthLayer_partA<<< gridSizeSixthLayer, blockSizeSixthLayerA>>>(Layer6_Neurons_GPU,
                        Layer6_Weights_GPU,
                        Layer7_Neurons_GPU,
                        Layer6_Mean_GPU,
                        Layer6_StanDev_GPU,
                        Layer6_Gamma_GPU,
                        Layer6_Beta_GPU
                    );
                    
    dim3 blockSizeSixthLayerB(32, 24);
    executeSixthLayer_partB<<< gridSizeSixthLayer, blockSizeSixthLayerB>>>(Layer6_Neurons_GPU,
                        Layer6_Weights_GPU,
                        Layer7_Neurons_GPU,
                        Layer6_Mean_GPU,
                        Layer6_StanDev_GPU,
                        Layer6_Gamma_GPU,
                        Layer6_Beta_GPU
                    );
    
    dim3 blockSizeSixthLayerC(24, 32);
    executeSixthLayer_partC<<< gridSizeSixthLayer, blockSizeSixthLayerC>>>(Layer6_Neurons_GPU,
                        Layer6_Weights_GPU,
                        Layer7_Neurons_GPU,
                        Layer6_Mean_GPU,
                        Layer6_StanDev_GPU,
                        Layer6_Gamma_GPU,
                        Layer6_Beta_GPU
                    );

    
    dim3 blockSizeSixthLayerD(24, 24);
    executeSixthLayer_partD<<< gridSizeSixthLayer, blockSizeSixthLayerD>>>(Layer6_Neurons_GPU,
                        Layer6_Weights_GPU,
                        Layer7_Neurons_GPU,
                        Layer6_Mean_GPU,
                        Layer6_StanDev_GPU,
                        Layer6_Gamma_GPU,
                        Layer6_Beta_GPU
                    );

    hipFree(Layer6_Weights_GPU);
    hipFree(Layer6_Mean_GPU);
    hipFree(Layer6_StanDev_GPU);
    hipFree(Layer6_Gamma_GPU);
    hipFree(Layer6_Beta_GPU);
}

void Read_SixthLayer_Data(double *Layer6_Weights_CPU,
    double * Layer6_Mean_CPU,
    double * Layer6_StanDev_CPU,
    double * Layer6_Gamma_CPU,
    double * Layer6_Beta_CPU
){
    read_File("data/SixthLayer/weightsNorm.txt", Layer6_Weights_CPU);
    read_File("data/SixthLayer/Sixth_Layer_Mean.txt", Layer6_Mean_CPU);
    read_File("data/SixthLayer/Sixth_Layer_StanDev.txt", Layer6_StanDev_CPU);
    read_File("data/SixthLayer/Sixth_Layer_Gamma.txt", Layer6_Gamma_CPU);
    read_File("data/SixthLayer/Sixth_Layer_Beta.txt", Layer6_Beta_CPU);
}

void read_File(const char * input_FileName, double * input_values){

    FILE *fp = fopen(input_FileName, "r");
    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;

    while ((read = getline(&line, &len, fp)) != -1)
        input_values[counter++] = atof(line);
    fclose(fp);
}

void read_Input_File(const char * inputFileName, double * Layer1_Neurons_CPU){
    FILE *fp = fopen(inputFileName, "r");

    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;
    int index = 0;
    int lastRow = 0;

    while ((read = getline(&line, &len, fp)) != -1) {
        Layer1_Neurons_CPU[counter++] = atof(line);
        index++;
        // handle padding
        if (index == 224){
            Layer1_Neurons_CPU[counter++] = 0;
            index = 0;
            lastRow++;
            if(lastRow == 224){
                lastRow = 0;
                int temp = 0;
                while (temp < 225) {
                    Layer1_Neurons_CPU[counter++] = 0;
                    temp++;
                }
            }
        }
    }
    fclose(fp);
}