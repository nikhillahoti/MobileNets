#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>

#include "MobileNets_kernel.cu"

#define INPUT_LAYER_SIZE 225 * 225 * 3
#define FIRST_LAYER_WEIGHT_SIZE 32 * 3 * 3 * 3
#define FIRST_LAYER_OUTPUT_SIZE 114 * 114 * 32
#define FIRST_LAYER_CHANNELS 32

#define SECOND_LAYER_WEIGHT_SIZE 32 * 3 * 3
#define SECOND_LAYER_OUTPUT_SIZE 112 * 112 * 32
#define SECOND_LAYER_CHANNELS 32

#define THIRD_LAYER_WEIGHT_SIZE 64 * 32
#define THIRD_LAYER_OUTPUT_SIZE 113 * 113 * 64
#define THIRD_LAYER_CHANNELS 64

#define FOURTH_LAYER_WEIGHT_SIZE 3 * 3 * 64
#define FOURTH_LAYER_OUTPUT_SIZE 56 * 56 * 64
#define FOURTH_LAYER_CHANNELS 64

// Function declarations
void NeuralNetwork();
void read_File(const char * weightFileName, double *Layer1_Weights_CPU);
void read_Input_File(const char * inputFileName, double *Layer1_Neurons_CPU);

void Read_First_Layer_Data(double * Layer1_Neurons_CPU,
    double * Layer1_Weights_CPU,
    double * Layer1_Mean_CPU,
    double * Layer1_StanDev_CPU,
    double * Layer1_Gamma_CPU,
    double * Layer1_Beta_CPU
);

void Execute_First_Layer(double * Layer2_Neurons_GPU);

void Read_SecondLayer_Data(double *Layer1_Weights_CPU,
    double *Layer2_Mean_CPU,
    double *Layer2_StanDev_CPU,
    double *Layer2_Gamma_CPU,
    double *Layer2_Beta_CPU
);

void Execute_Second_Layer(
    double * Layer2_Neurons_GPU,
    double * Layer3_Neurons_GPU
);

void Read_ThirdLayer_Data(double *Layer3_Weights_CPU,
    double * Layer3_Mean_CPU,
    double * Layer3_StanDev_CPU,
    double * Layer3_Gamma_CPU,
    double * Layer3_Beta_CPU
);
void Execute_Third_Layer(
    double * Layer3_Neurons_GPU,
    double * Layer4_Neurons_GPU
);

void Read_FourthLayer_Data(double *Layer4_Weights_CPU,
    double * Layer4_Mean_CPU,
    double * Layer4_StanDev_CPU,
    double * Layer4_Gamma_CPU,
    double * Layer4_Beta_CPU
);
void Execute_Fourth_Layer(
    double * Layer4_Neurons_GPU,
    double * Layer5_Neurons_GPU
);

int main(){
    NeuralNetwork();
}

void NeuralNetwork(){
    FILE *fOutput;
    int value;

    /* ************************************************ FIRST LAYER ******************************************************** */
    double *Layer2_Neurons_GPU = NULL; 
    hipMalloc((void**) &Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);

    Execute_First_Layer(Layer2_Neurons_GPU);

    // Saving output of the first layer: Initially Not Saved
    bool SAVE_FIRST_LAYER_WEIGHTS = false;
    if(SAVE_FIRST_LAYER_WEIGHTS){
        
        double *Layer2_Neurons_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer2_Neurons_CPU, Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/FirstLayer/output.txt", "w");
        value = FIRST_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer2_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer2_Neurons_CPU);
    }
    
    printf("\n Layer 1 Execution complete !!!");
    /* ************************************************ FIRST LAYER COMPLETE *********************************************** */

    /* ************************************************ SECOND LAYER ******************************************************** */
    double *Layer3_Neurons_GPU;
    hipMalloc((void**) &Layer3_Neurons_GPU, sizeof(double) * SECOND_LAYER_OUTPUT_SIZE);

    Execute_Second_Layer(Layer2_Neurons_GPU, Layer3_Neurons_GPU);

    bool SAVE_SECOND_LAYER_WEIGHTS = false;
    if(SAVE_SECOND_LAYER_WEIGHTS){
        
        double * Layer3_Neurons_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer3_Neurons_CPU, Layer3_Neurons_GPU, sizeof(double) * SECOND_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/SecondLayer/output.txt", "w");
        value = SECOND_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer3_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer3_Neurons_CPU);
    }

    printf("\n Layer 2 Execution complete !!!");
    /* ************************************************ SECOND LAYER COMPLETE *********************************************** */

    /* ************************************************ THIRD LAYER ******************************************************** */
    double *Layer4_Neurons_GPU;
    hipMalloc((void**) &Layer4_Neurons_GPU, sizeof(double) * THIRD_LAYER_OUTPUT_SIZE);

    Execute_Third_Layer(Layer3_Neurons_GPU, Layer4_Neurons_GPU);

    bool SAVE_THIRD_LAYER_WEIGHTS = true;
    if(SAVE_THIRD_LAYER_WEIGHTS){
        double * Layer4_Neurons_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer4_Neurons_CPU, Layer4_Neurons_GPU, sizeof(double) * THIRD_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/ThirdLayer/output.txt", "w");
        value = THIRD_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer4_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer4_Neurons_CPU);
    }

    printf("\n Layer 3 Execution complete !!!");
    /* ************************************************ THIRD LAYER COMPLETE *********************************************** */

    /* ************************************************ FOURTH LAYER ******************************************************** */
    double *Layer5_Neurons_GPU;
    hipMalloc((void**) &Layer5_Neurons_GPU, sizeof(double) * FOURTH_LAYER_OUTPUT_SIZE);

    Execute_Fourth_Layer(Layer4_Neurons_GPU, Layer5_Neurons_GPU);

    bool SAVE_FOURTH_LAYER_WEIGHTS = true;
    if(SAVE_FOURTH_LAYER_WEIGHTS){
        double * Layer5_Neurons_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_OUTPUT_SIZE);
        hipMemcpy(Layer5_Neurons_CPU, Layer5_Neurons_GPU, sizeof(double) * FOURTH_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        // Logic to save into the file to verify the results
        fOutput = fopen("data/FourthLayer/output.txt", "w");
        value = FOURTH_LAYER_OUTPUT_SIZE;
        for(int i = 0 ; i < value ; i++){
            fprintf (fOutput, "%0.6lf\n", Layer5_Neurons_CPU[i]);
        }
        fclose(fOutput);

        free(Layer5_Neurons_CPU);
    }

    printf("\n Layer 4 Execution complete !!!");
    /* ************************************************ FOURTH LAYER COMPLETE *********************************************** */

    printf("\n\n Processing Done !!! \n\n");

    hipFree(Layer2_Neurons_GPU);
    hipFree(Layer3_Neurons_GPU);
    hipFree(Layer4_Neurons_GPU);
}

void Execute_First_Layer(double *Layer2_Neurons_GPU)
{
    double *Layer1_Neurons_CPU = (double *) malloc(sizeof(double) * INPUT_LAYER_SIZE);
    double *Layer1_Weights_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);
    double *Layer1_Mean_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double *Layer1_StanDev_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double *Layer1_Gamma_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double *Layer1_Beta_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);

    Read_First_Layer_Data(
        Layer1_Neurons_CPU,
        Layer1_Weights_CPU,        
        Layer1_Mean_CPU,
        Layer1_StanDev_CPU,
        Layer1_Gamma_CPU,
        Layer1_Beta_CPU
    );

    // Copy memory from Host to Kernel
    double *Layer1_Weights_GPU,
           *Layer1_Neurons_GPU,
           *Layer1_Mean_GPU,
           *Layer1_StanDev_GPU,
           *Layer1_Gamma_GPU,
           *Layer1_Beta_GPU;

    hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(double) * INPUT_LAYER_SIZE);
    hipMalloc((void**) &Layer1_Weights_GPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer1_Mean_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_StanDev_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_Gamma_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_Beta_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);

    hipMemcpy(Layer1_Neurons_GPU, Layer1_Neurons_CPU, sizeof(double) * INPUT_LAYER_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Weights_GPU, Layer1_Weights_CPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Mean_GPU, Layer1_Mean_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_StanDev_GPU, Layer1_StanDev_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Gamma_GPU, Layer1_Gamma_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Beta_GPU, Layer1_Beta_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer1_Neurons_CPU);
    free(Layer1_Weights_CPU);
    free(Layer1_Mean_CPU);
    free(Layer1_StanDev_CPU);
    free(Layer1_Gamma_CPU);
    free(Layer1_Beta_CPU);

    // Kernel Launch
    dim3 gridSizeA(32, 3, 3);
    dim3 blockSizeA(32,32);

    executeFirstLayer_partA<<< gridSizeA, blockSizeA>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );
    
    dim3 gridSizeB(32, 7);
    dim3 blockSizeB(16, 16);

    executeFirstLayer_partB<<< gridSizeB, blockSizeB>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );

    dim3 gridSizeC(32, 6);
    dim3 blockSizeC(16, 16);

    executeFirstLayer_partC<<< gridSizeC, blockSizeC>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );

    hipDeviceSynchronize();

    // First Layer GPU Memory Free
    hipFree(Layer1_Neurons_GPU);
    hipFree(Layer1_Weights_GPU);
    hipFree(Layer1_Mean_GPU);
    hipFree(Layer1_StanDev_GPU);
    hipFree(Layer1_Gamma_GPU);
    hipFree(Layer1_Beta_GPU);
}

void Read_First_Layer_Data(
    double * Layer1_Neurons_CPU,
    double * Layer1_Weights_CPU,
    double * Layer1_Mean_CPU,
    double * Layer1_StanDev_CPU,
    double * Layer1_Gamma_CPU,
    double * Layer1_Beta_CPU
){
    read_Input_File("data/FirstLayer/InputFiles/inputNorm.txt", Layer1_Neurons_CPU);
    read_File("data/FirstLayer/weightsNorm.txt", Layer1_Weights_CPU);
    read_File("data/FirstLayer/First_Layer_Mean.txt", Layer1_Mean_CPU);
    read_File("data/FirstLayer/First_Layer_StanDev.txt", Layer1_StanDev_CPU);
    read_File("data/FirstLayer/First_Layer_Gamma.txt", Layer1_Gamma_CPU);
    read_File("data/FirstLayer/First_Layer_Beta.txt", Layer1_Beta_CPU);
}

void Execute_Second_Layer(
    double * Layer2_Neurons_GPU,
    double * Layer3_Neurons_GPU
)
{
    double * Layer2_Weights_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_WEIGHT_SIZE);
    double * Layer2_Mean_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);
    double * Layer2_StanDev_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);
    double * Layer2_Gamma_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);
    double * Layer2_Beta_CPU = (double *) malloc(sizeof(double) * SECOND_LAYER_CHANNELS);


    Read_SecondLayer_Data(Layer2_Weights_CPU,
                        Layer2_Mean_CPU,
                        Layer2_StanDev_CPU,
                        Layer2_Gamma_CPU,
                        Layer2_Beta_CPU
    );
    
    double *Layer2_Weights_GPU,
           *Layer2_Mean_GPU,
           *Layer2_StanDev_GPU,
           *Layer2_Gamma_GPU,
           *Layer2_Beta_GPU;;

    hipMalloc((void**) &Layer2_Weights_GPU, sizeof(double) * SECOND_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer2_Mean_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);
    hipMalloc((void**) &Layer2_StanDev_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);
    hipMalloc((void**) &Layer2_Gamma_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);
    hipMalloc((void**) &Layer2_Beta_GPU, sizeof(double) * SECOND_LAYER_CHANNELS);

    hipMemcpy(Layer2_Weights_GPU, Layer2_Weights_CPU, sizeof(double) * SECOND_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Mean_GPU, Layer2_Mean_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_StanDev_GPU, Layer2_StanDev_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Gamma_GPU, Layer2_Gamma_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Beta_GPU, Layer2_Beta_CPU, sizeof(double) * SECOND_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer2_Weights_CPU);
    free(Layer2_Mean_CPU);
    free(Layer2_StanDev_CPU);
    free(Layer2_Gamma_CPU);
    free(Layer2_Beta_CPU);

    dim3 gridSizeA(32, 3, 3);
    dim3 blockSizeA(32,32);
    executeSecondLayer_partA<<< gridSizeA, blockSizeA>>>(Layer2_Neurons_GPU,
                                            Layer2_Weights_GPU,
                                            Layer3_Neurons_GPU,
                                            Layer2_Mean_GPU,
                                            Layer2_StanDev_GPU,
                                            Layer2_Gamma_GPU,
                                            Layer2_Beta_GPU
    );

    dim3 gridSizeB(32, 7);
    dim3 blockSizeB(16, 16);
    executeSecondLayer_partB<<< gridSizeB, blockSizeB>>>(Layer2_Neurons_GPU,
                                            Layer2_Weights_GPU,
                                            Layer3_Neurons_GPU,
                                            Layer2_Mean_GPU,
                                            Layer2_StanDev_GPU,
                                            Layer2_Gamma_GPU,
                                            Layer2_Beta_GPU
    );

    dim3 gridSizeC(32, 6);
    dim3 blockSizeC(16, 16);
    executeSecondLayer_partC<<< gridSizeC, blockSizeC>>>(Layer2_Neurons_GPU,
                                            Layer2_Weights_GPU,
                                            Layer3_Neurons_GPU,
                                            Layer2_Mean_GPU,
                                            Layer2_StanDev_GPU,
                                            Layer2_Gamma_GPU,
                                            Layer2_Beta_GPU
    );

    hipFree(Layer2_Weights_GPU);    
    hipFree(Layer2_Mean_GPU);
    hipFree(Layer2_StanDev_GPU);
    hipFree(Layer2_Gamma_GPU);
    hipFree(Layer2_Beta_GPU);
}

void Read_SecondLayer_Data(double *Layer2_Weights_CPU,
    double * Layer2_Mean_CPU,
    double * Layer2_StanDev_CPU,
    double * Layer2_Gamma_CPU,
    double * Layer2_Beta_CPU
){
    read_File("data/SecondLayer/weightsNorm.txt", Layer2_Weights_CPU);
    read_File("data/SecondLayer/Second_Layer_Mean.txt", Layer2_Mean_CPU);
    read_File("data/SecondLayer/Second_Layer_StanDev.txt", Layer2_StanDev_CPU);
    read_File("data/SecondLayer/Second_Layer_Gamma.txt", Layer2_Gamma_CPU);
    read_File("data/SecondLayer/Second_Layer_Beta.txt", Layer2_Beta_CPU);
}

void Execute_Third_Layer(
    double * Layer3_Neurons_GPU,
    double * Layer4_Neurons_GPU
){
    double * Layer3_Weights_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_WEIGHT_SIZE);
    double * Layer3_Mean_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);
    double * Layer3_StanDev_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);
    double * Layer3_Gamma_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);
    double * Layer3_Beta_CPU = (double *) malloc(sizeof(double) * THIRD_LAYER_CHANNELS);

    Read_ThirdLayer_Data(Layer3_Weights_CPU,
                Layer3_Mean_CPU,
                Layer3_StanDev_CPU,
                Layer3_Gamma_CPU,
                Layer3_Beta_CPU
    );

    double *Layer3_Weights_GPU,
           *Layer3_Mean_GPU,
           *Layer3_StanDev_GPU,
           *Layer3_Gamma_GPU,
           *Layer3_Beta_GPU;

    hipMalloc((void**) &Layer3_Weights_GPU, sizeof(double) * THIRD_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer3_Mean_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);
    hipMalloc((void**) &Layer3_StanDev_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);
    hipMalloc((void**) &Layer3_Gamma_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);
    hipMalloc((void**) &Layer3_Beta_GPU, sizeof(double) * THIRD_LAYER_CHANNELS);

    hipMemcpy(Layer3_Weights_GPU, Layer3_Weights_CPU, sizeof(double) * THIRD_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Mean_GPU, Layer3_Mean_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_StanDev_GPU, Layer3_StanDev_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Gamma_GPU, Layer3_Gamma_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer3_Beta_GPU, Layer3_Beta_CPU, sizeof(double) * THIRD_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer3_Weights_CPU);
    free(Layer3_Mean_CPU);
    free(Layer3_StanDev_CPU);
    free(Layer3_Gamma_CPU);
    free(Layer3_Beta_CPU);
    
    // Execution of the Third Layer
    dim3 gridSizeThirdLayerA(64, 3, 3);
    dim3 blockSizeThirdLayerA(32,32);
    executeThirdLayer_partA<<< gridSizeThirdLayerA, blockSizeThirdLayerA>>>(Layer3_Neurons_GPU,
                        Layer3_Weights_GPU,
                        Layer4_Neurons_GPU,
                        Layer3_Mean_GPU,
                        Layer3_StanDev_GPU,
                        Layer3_Gamma_GPU,
                        Layer3_Beta_GPU
    );

    dim3 gridSizeThirdLayerB(64, 7);
    dim3 blockSizeThirdLayerB(16, 16);
    executeThirdLayer_partB<<< gridSizeThirdLayerB, blockSizeThirdLayerB>>>(Layer3_Neurons_GPU,
                        Layer3_Weights_GPU,
                        Layer4_Neurons_GPU,
                        Layer3_Mean_GPU,
                        Layer3_StanDev_GPU,
                        Layer3_Gamma_GPU,
                        Layer3_Beta_GPU
    );

    dim3 gridSizeThirdLayerC(64, 6);
    dim3 blockSizeThirdLayerC(16, 16);
    executeThirdLayer_partC<<< gridSizeThirdLayerC, blockSizeThirdLayerC>>>(Layer3_Neurons_GPU,
                        Layer3_Weights_GPU,
                        Layer4_Neurons_GPU,
                        Layer3_Mean_GPU,
                        Layer3_StanDev_GPU,
                        Layer3_Gamma_GPU,
                        Layer3_Beta_GPU
    );

    hipDeviceSynchronize();

    hipFree(Layer3_Weights_GPU);
    hipFree(Layer3_Mean_GPU);
    hipFree(Layer3_StanDev_GPU);
    hipFree(Layer3_Gamma_GPU);
    hipFree(Layer3_Beta_GPU);
}

void Read_ThirdLayer_Data(double *Layer3_Weights_CPU,
    double * Layer3_Mean_CPU,
    double * Layer3_StanDev_CPU,
    double * Layer3_Gamma_CPU,
    double * Layer3_Beta_CPU
){
    read_File("data/ThirdLayer/weightsNorm.txt", Layer3_Weights_CPU);
    read_File("data/ThirdLayer/Third_Layer_Mean.txt", Layer3_Mean_CPU);
    read_File("data/ThirdLayer/Third_Layer_StanDev.txt", Layer3_StanDev_CPU);
    read_File("data/ThirdLayer/Third_Layer_Gamma.txt", Layer3_Gamma_CPU);
    read_File("data/ThirdLayer/Third_Layer_Beta.txt", Layer3_Beta_CPU);
}

void Execute_Fourth_Layer(
    double * Layer4_Neurons_GPU,
    double * Layer5_Neurons_GPU
){  
    double * Layer4_Weights_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_WEIGHT_SIZE);
    double * Layer4_Mean_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);
    double * Layer4_StanDev_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);
    double * Layer4_Gamma_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);
    double * Layer4_Beta_CPU = (double *) malloc(sizeof(double) * FOURTH_LAYER_CHANNELS);

    Read_FourthLayer_Data(Layer4_Weights_CPU,
                    Layer4_Mean_CPU,
                    Layer4_StanDev_CPU,
                    Layer4_Gamma_CPU,
                    Layer4_Beta_CPU
    );
    
    double *Layer4_Weights_GPU,
           *Layer4_Mean_GPU,
           *Layer4_StanDev_GPU,
           *Layer4_Gamma_GPU,
           *Layer4_Beta_GPU;

    hipMalloc((void**) &Layer4_Weights_GPU, sizeof(double) * FOURTH_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer4_Mean_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer4_StanDev_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer4_Gamma_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);
    hipMalloc((void**) &Layer4_Beta_GPU, sizeof(double) * FOURTH_LAYER_CHANNELS);

    hipMemcpy(Layer4_Weights_GPU, Layer4_Weights_CPU, sizeof(double) * FOURTH_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Mean_GPU, Layer4_Mean_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_StanDev_GPU, Layer4_StanDev_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Gamma_GPU, Layer4_Gamma_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer4_Beta_GPU, Layer4_Beta_CPU, sizeof(double) * FOURTH_LAYER_CHANNELS, hipMemcpyHostToDevice);

    free(Layer4_Weights_CPU);
    free(Layer4_Mean_CPU);
    free(Layer4_StanDev_CPU);
    free(Layer4_Gamma_CPU);
    free(Layer4_Beta_CPU);

    dim3 gridSizeFourthLayer(64);
    dim3 blockSizeFourthLayerA(32,32);
    executeFourthLayer_partA<<< gridSizeFourthLayer, blockSizeFourthLayerA>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    dim3 blockSizeFourthLayerB(32, 24);
    executeFourthLayer_partB<<< gridSizeFourthLayer, blockSizeFourthLayerB>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    
    dim3 blockSizeFourthLayerC(24, 32);
    executeFourthLayer_partC<<< gridSizeFourthLayer, blockSizeFourthLayerC>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    
    dim3 blockSizeFourthLayerD(24, 24);
    executeFourthLayer_partD<<< gridSizeFourthLayer, blockSizeFourthLayerD>>>(Layer4_Neurons_GPU,
                        Layer4_Weights_GPU,
                        Layer5_Neurons_GPU,
                        Layer4_Mean_GPU,
                        Layer4_StanDev_GPU,
                        Layer4_Gamma_GPU,
                        Layer4_Beta_GPU
                    );

    hipFree(Layer4_Weights_GPU);
    hipFree(Layer4_Mean_GPU);
    hipFree(Layer4_StanDev_GPU);
    hipFree(Layer4_Gamma_GPU);
    hipFree(Layer4_Beta_GPU);
}

void Read_FourthLayer_Data(double *Layer4_Weights_CPU,
    double * Layer4_Mean_CPU,
    double * Layer4_StanDev_CPU,
    double * Layer4_Gamma_CPU,
    double * Layer4_Beta_CPU
){
    read_File("data/FourthLayer/weightsNorm.txt", Layer4_Weights_CPU);
    read_File("data/FourthLayer/Fourth_Layer_Mean.txt", Layer4_Mean_CPU);
    read_File("data/FourthLayer/Fourth_Layer_StanDev.txt", Layer4_StanDev_CPU);
    read_File("data/FourthLayer/Fourth_Layer_Gamma.txt", Layer4_Gamma_CPU);
    read_File("data/FourthLayer/Fourth_Layer_Beta.txt", Layer4_Beta_CPU);
}

void read_File(const char * input_FileName, double * input_values){

    FILE *fp = fopen(input_FileName, "r");
    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;

    while ((read = getline(&line, &len, fp)) != -1)
        input_values[counter++] = atof(line);
    fclose(fp);
}

void read_Input_File(const char * inputFileName, double * Layer1_Neurons_CPU){
    FILE *fp = fopen(inputFileName, "r");

    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;
    int index = 0;
    int lastRow = 0;

    while ((read = getline(&line, &len, fp)) != -1) {
        Layer1_Neurons_CPU[counter++] = atof(line);
        index++;
        // handle padding
        if (index == 224){
            Layer1_Neurons_CPU[counter++] = 0;
            index = 0;
            lastRow++;
            if(lastRow == 224){
                lastRow = 0;
                int temp = 0;
                while (temp < 225) {
                    Layer1_Neurons_CPU[counter++] = 0;
                    temp++;
                }
            }
        }
    }
    fclose(fp);
}