#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>

#include "MobileNets_kernel.cu"

#define INPUT_LAYER_SIZE 225 * 225 * 3
#define FIRST_LAYER_WEIGHT_SIZE 32 * 3 * 3 * 3
#define FIRST_LAYER_OUTPUT_SIZE 114 * 114 * 32
#define FIRST_LAYER_CHANNELS 32

// Function declarations
void Read_First_Layer_Data(double * Layer1_Neurons_CPU,
     double * Layer1_Weights_CPU,
     double * Layer1_Mean_CPU,
     double * Layer1_StanDev_CPU,
     double * Layer1_Gamma_CPU,
     double * Layer1_Beta_CPU
);

void NeuralNetwork();
void read_File(const char * weightFileName, double *Layer1_Weights_CPU);
void read_Input_File(const char * inputFileName, double *Layer1_Neurons_CPU);

int main(){
    NeuralNetwork();
}

void NeuralNetwork(){
    // Reading the input layer data
    double * Layer1_Neurons_CPU = (double *) malloc(sizeof(double) * INPUT_LAYER_SIZE);
    double * Layer1_Weights_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);
    double * Layer1_Mean_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double * Layer1_StanDev_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double * Layer1_Gamma_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);
    double * Layer1_Beta_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_CHANNELS);

    Read_First_Layer_Data(Layer1_Neurons_CPU,
                Layer1_Weights_CPU,
                Layer1_Mean_CPU,
                Layer1_StanDev_CPU,
                Layer1_Gamma_CPU,
                Layer1_Beta_CPU
    );

    // Allocating memory for Output Matrix
    double * Layer2_Neurons_CPU = (double *) malloc(sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);

    // Copy memory from Host to Kernel
    double *Layer1_Weights_GPU,
           *Layer1_Neurons_GPU,
           *Layer2_Neurons_GPU,
           *Layer1_Mean_GPU,
           *Layer1_StanDev_GPU,
           *Layer1_Gamma_GPU,
           *Layer1_Beta_GPU;

    hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(double) * INPUT_LAYER_SIZE);
    hipMalloc((void**) &Layer1_Weights_GPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE);
    hipMalloc((void**) &Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE);
    hipMalloc((void**) &Layer1_Mean_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_StanDev_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_Gamma_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);
    hipMalloc((void**) &Layer1_Beta_GPU, sizeof(double) * FIRST_LAYER_CHANNELS);

    hipMemcpy(Layer1_Neurons_GPU, Layer1_Neurons_CPU, sizeof(double) * INPUT_LAYER_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Weights_GPU, Layer1_Weights_CPU, sizeof(double) * FIRST_LAYER_WEIGHT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer2_Neurons_GPU, Layer2_Neurons_CPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Mean_GPU, Layer1_Mean_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_StanDev_GPU, Layer1_StanDev_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Gamma_GPU, Layer1_Gamma_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);
    hipMemcpy(Layer1_Beta_GPU, Layer1_Beta_CPU, sizeof(double) * FIRST_LAYER_CHANNELS, hipMemcpyHostToDevice);


    // Kernel Launch
    dim3 gridSizeA(32, 3, 3);
    dim3 blockSizeA(32,32);

    executeFirstLayer_partA<<< gridSizeA, blockSizeA>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );

    dim3 gridSizeB(32, 7);
    dim3 blockSizeB(16, 16);

    executeFirstLayer_partB<<< gridSizeB, blockSizeB>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );

    dim3 gridSizeC(32, 6);
    dim3 blockSizeC(16, 16);

    executeFirstLayer_partC<<< gridSizeC, blockSizeC>>>(Layer1_Neurons_GPU,
                        Layer1_Weights_GPU,
                        Layer2_Neurons_GPU,
                        Layer1_Mean_GPU,
                        Layer1_StanDev_GPU,
                        Layer1_Gamma_GPU,
                        Layer1_Beta_GPU
                    );


    // Get back the data from the kernel to the host
    hipMemcpy(Layer2_Neurons_CPU, Layer2_Neurons_GPU, sizeof(double) * FIRST_LAYER_OUTPUT_SIZE, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Logic to save into the file to verify the results
    FILE * fOutput = fopen("data/FirstLayer/output.txt", "w");
    int value = FIRST_LAYER_OUTPUT_SIZE;
    for(int i = 0 ; i < value ; i++){
        fprintf (fOutput, "%0.7lf\n", Layer2_Neurons_CPU[i]);
    }
    fclose(fOutput);

    printf("\n\n Processing Done !!! \n\n");

    // Free the memory at the end
    free(Layer1_Neurons_CPU);
    free(Layer1_Weights_CPU);
    free(Layer2_Neurons_CPU);
    free(Layer1_Mean_CPU);
    free(Layer1_StanDev_CPU);
    free(Layer1_Gamma_CPU);
    free(Layer1_Beta_CPU);

    hipFree(Layer1_Neurons_GPU);
    hipFree(Layer1_Weights_GPU);
    hipFree(Layer2_Neurons_GPU);
    hipFree(Layer1_Mean_GPU);
    hipFree(Layer1_StanDev_GPU);
    hipFree(Layer1_Gamma_GPU);
    hipFree(Layer1_Beta_GPU);
}

void Read_First_Layer_Data(double * Layer1_Neurons_CPU,
    double * Layer1_Weights_CPU,
    double * Layer1_Mean_CPU,
    double * Layer1_StanDev_CPU,
    double * Layer1_Gamma_CPU,
    double * Layer1_Beta_CPU
){
    read_Input_File("data/FirstLayer/Input_File.txt", Layer1_Neurons_CPU);
    read_File("data/FirstLayer/First_Layer_Weights.txt", Layer1_Weights_CPU);
    read_File("data/FirstLayer/First_Layer_Mean.txt", Layer1_Mean_CPU);
    read_File("data/FirstLayer/First_Layer_StanDev.txt", Layer1_StanDev_CPU);
    read_File("data/FirstLayer/First_Layer_Gamma.txt", Layer1_Gamma_CPU);
    read_File("data/FirstLayer/First_Layer_Beta.txt", Layer1_Beta_CPU);
}


void read_File(const char * input_FileName, double * input_values){

    FILE *fp = fopen(input_FileName, "r");
    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;

    while ((read = getline(&line, &len, fp)) != -1)
        input_values[counter++] = atof(line);
    fclose(fp);
}

void read_Input_File(const char * inputFileName, double * Layer1_Neurons_CPU){
    FILE *fp = fopen(inputFileName, "r");

    if (fp == NULL){
        printf("\n No input file present at the location \n");
        return;
    }

    int counter = 0;
    ssize_t read;
    char * line = NULL;
    size_t len = 1000;
    int index = 0;
    int lastRow = 0;

    while ((read = getline(&line, &len, fp)) != -1) {
        Layer1_Neurons_CPU[counter++] = atof(line);
        index++;
        // handle padding
        if (index == 224){
            Layer1_Neurons_CPU[counter++] = 0;
            index = 0;
            lastRow++;
            if(lastRow == 224){
                lastRow = 0;
                int temp = 0;
                while (temp < 225) {
                    Layer1_Neurons_CPU[counter++] = 0;
                    temp++;
                }
            }
        }
    }

    printf("\n Total characters read ---> %d\n", counter);
    fclose(fp);
}
